#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <granular/DataStructs.h>

inline __device__ void cleanUpForces(unsigned int thisClump, sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData) {
    // Actually, h should be JITCed into the kernel itself
    granData->h2aX[thisClump] = 0;
    granData->h2aY[thisClump] = 0;
    granData->h2aZ[thisClump] = 0;
}

inline __device__ void applyGravity(unsigned int thisClump, sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData) {
    // Actually, l should be JITCed into the kernel itself
    granData->h2aX[thisClump] += simParams->h * simParams->h * simParams->Gx / simParams->l;
    granData->h2aY[thisClump] += simParams->h * simParams->h * simParams->Gy / simParams->l;
    granData->h2aZ[thisClump] += simParams->h * simParams->h * simParams->Gz / simParams->l;
}

__global__ void deriveClumpForces(sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData) {
    unsigned int thisClump = blockIdx.x * blockDim.x + threadIdx.x;
    if (thisClump < simParams->nClumpBodies) {
        cleanUpForces(thisClump, simParams, granData);
        applyGravity(thisClump, simParams, granData);
    }
}
