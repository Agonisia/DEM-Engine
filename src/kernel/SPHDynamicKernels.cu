#include "hip/hip_runtime.h"
// All Dynamic CUDA kernels for SPH in gpu-physics

#include <sph/datastruct.h>
#include <kernel/CUDAMathHelpers.cu>
// =================================================================================================================
// ========================================= START of Dynamic kernels ==============================================
// =================================================================================================================

// =================================================================================================================
// Dynamic 1st Pass, this pass compute the contact force of each contact pair
// the computed force will be filled in contact_force field in each contact pair
// =================================================================================================================
__global__ void dynamicStep1(int* pair_i_data,
                             int* pair_j_data,
                             float* rho_data,
                             float* pressure_data,
                             float3* col_acc_data,
                             float3* W_grad_data,
                             int n_col,
                             float kernel_h,
                             float m) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n_col) {
        return;
    }

    int i_idx = pair_i_data[idx];
    int j_idx = pair_j_data[idx];

    if (rho_data[i_idx] != 0 && rho_data[j_idx] != 0) {
        float coe = m * ((pressure_data[i_idx] / (rho_data[i_idx] * rho_data[i_idx])) +
                         (pressure_data[j_idx] / (rho_data[j_idx] * rho_data[j_idx])));
        col_acc_data[idx] = -coe * W_grad_data[idx];
    }
}

__global__ void dynamicStep2(int* pair_i_data_reduced, float3* col_acc_data_reduced, float3* acc_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) {
        return;
    }

    acc_data[pair_i_data_reduced[idx]] = col_acc_data_reduced[idx];
}

__global__ void dynamicStep3(float3* col_acc_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) {
        return;
    }

    col_acc_data[idx].x = -col_acc_data[idx].x;
    col_acc_data[idx].y = -col_acc_data[idx].y;
    col_acc_data[idx].z = -col_acc_data[idx].z;
}

__global__ void dynamicStep4(int* pair_j_data_reduced, float3* col_acc_data_reduced, float3* acc_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) {
        return;
    }

    acc_data[pair_j_data_reduced[idx]] = col_acc_data_reduced[idx];
}

// ===========================================

__global__ void dynamicStep5(float3* pos_data,
                             float3* vel_data,
                             float3* acc_data,
                             char* fix_data,
                             int n,
                             float time_step,
                             float kernel_h) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) {
        return;
    }

    if (fix_data[idx] == 0) {
        float grav = -9.8f;
        acc_data[idx].z += grav;

        vel_data[idx].x += acc_data[idx].x * time_step;
        vel_data[idx].y += acc_data[idx].y * time_step;
        vel_data[idx].z += acc_data[idx].z * time_step;

        pos_data[idx].x += vel_data[idx].x * time_step;
        pos_data[idx].y += vel_data[idx].y * time_step;
        pos_data[idx].z += vel_data[idx].z * time_step;
    } else {
        acc_data[idx] = make_float3(0.f, 0.f, 0.f);
    }

    __syncthreads();
}

/*
// =================================================================================================================
// Dynamic 3rd pass, this pass is intended to copy all reduced data into global memory gpu_acc
// TODO: reconsider the necessity of existence of this kernel
// =================================================================================================================
__global__ void dynamicStep3(int* key, float* x_reduced, float* y_reduced, float* z_reduced, int n, vector3* gpu_acc) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) {
        return;
    }
    __syncthreads();

    gpu_acc[key[idx]].x = x_reduced[idx];
    gpu_acc[key[idx]].y = y_reduced[idx];
    gpu_acc[key[idx]].z = z_reduced[idx];
}

// =================================================================================================================
// Dynamic 4th pass, the actual integration pass
// this pass will integrate gpu_acc to gpu_vel to gpu_pos
// and push simulation 1 step forward
// =================================================================================================================
__global__ void dynamicStep4(vector3* gpu_pos,
                             vector3* gpu_vel,
                             vector3* gpu_acc,
                             char* gpu_fix,
                             int gpu_n,
                             float time_step,
                             float kernel_h) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= gpu_n) {
        return;
    }

    if (gpu_fix[idx] == 0) {
        float grav = -9.8f;
        gpu_acc[idx].z = gpu_acc[idx].z + grav;

        gpu_vel[idx].x = gpu_vel[idx].x + gpu_acc[idx].x * time_step;
        gpu_vel[idx].y = gpu_vel[idx].y + gpu_acc[idx].y * time_step;
        gpu_vel[idx].z = gpu_vel[idx].z + gpu_acc[idx].z * time_step;

        gpu_pos[idx].x = gpu_pos[idx].x + gpu_vel[idx].x * time_step;
        gpu_pos[idx].y = gpu_pos[idx].y + gpu_vel[idx].y * time_step;
        gpu_pos[idx].z = gpu_pos[idx].z + gpu_vel[idx].z * time_step;
    }
    __syncthreads();

    gpu_acc[idx].x = 0.f;
    gpu_acc[idx].y = 0.f;
    gpu_acc[idx].z = 0.f;
}
*/

// =================================================================================================================
// ========================================= END of Dynamic kernels ================================================
// =================================================================================================================