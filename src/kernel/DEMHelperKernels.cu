#include "hip/hip_runtime.h"
// DEM device-side helper kernel collection
#include <helper_math.cuh>
#include <granular/DataStructs.h>
#include <granular/GranularDefines.h>

// inline __device__ double3 voxelID2LRFPosition
// inline __device__ voxelID_t position2VoxelID

// Sign function
template <typename T>
inline __device__ int sgn(T val) {
    return (T(0) < val) - (val < T(0));
}

// Integer division that rounds towards -infty
template <typename T1, typename T2>
inline __device__ T1 div_floor(T1 a, T2 b) {
    T1 res = a / b;
    T1 rem = a % b;
    // Correct division result downwards if up-rounding happened,
    // (for non-zero remainder of sign different than the divisor).
    T1 corr = (rem != 0 && ((rem < 0) != (b < 0)));
    return res - corr;
}

// Modulus that rounds towards -infty
template <typename T1, typename T2>
inline __device__ T1 mod_floor(T1 a, T2 b) {
    if (b < 0)  // you can check for b == 0 separately and do what you want
        return -mod_floor(-a, -b);
    T1 ret = a % b;
    if (ret < 0)
        ret += b;
    return ret;
}

// Chops a long ID (typically voxelID) into XYZ components
template <typename T1, typename T2>
inline __device__ void IDChopper(T1& X, T1& Y, T1& Z, T2& ID, unsigned char& nvXp2, unsigned char& nvYp2) {
    X = ID & (((T1)1 << nvXp2) - 1);  // & operation here equals modulo
    Y = (ID >> nvXp2) & (((T1)1 << nvYp2) - 1);
    Z = (ID) >> (nvXp2 + nvYp2);
}

// Packs XYZ components back to a long ID (typically voxelID)
template <typename T1, typename T2>
inline __device__ void IDPacker(T1& ID, T2& X, T2& Y, T2& Z, unsigned char& nvXp2, unsigned char& nvYp2) {
    ID = 0;
    ID += X;
    ID += Y << nvXp2;
    ID += Z << (nvXp2 + nvYp2);
}

template <typename T1, typename T2>
inline __device__ void applyOriQToVector3(T1& X, T1& Y, T1& Z) {
    // Now does nothing
}

template <typename T1>
inline __device__ T1 distSquared(T1 x1, T1 y1, T1 z1, T1 x2, T1 y2, T1 z2) {
    return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2);
}
