#include "hip/hip_runtime.h"
// DEM device-side helper kernel collection
#include <helper_math.cuh>
#include <granular/DataStructs.h>
#include <granular/GranularDefines.h>

// inline __device__ voxelID_t position2VoxelID

// Sign function
template <typename T1>
inline __device__ int sgn(const T1& val) {
    return (T1(0) < val) - (val < T1(0));
}

// Integer division that rounds towards -infty
template <typename T1, typename T2>
inline __device__ T1 div_floor(const T1& a, const T2& b) {
    T1 res = a / b;
    T1 rem = a % b;
    // Correct division result downwards if up-rounding happened,
    // (for non-zero remainder of sign different than the divisor).
    T1 corr = (rem != 0 && ((rem < 0) != (b < 0)));
    return res - corr;
}

// Modulus that rounds towards -infty
template <typename T1, typename T2>
inline __device__ T1 mod_floor(const T1& a, const T2& b) {
    if (b < 0)  // you can check for b == 0 separately and do what you want
        return -mod_floor(-a, -b);
    T1 ret = a % b;
    if (ret < 0)
        ret += b;
    return ret;
}

// Chops a long ID (typically voxelID) into XYZ components
template <typename T1, typename T2>
inline __device__ void IDChopper(T1& X,
                                 T1& Y,
                                 T1& Z,
                                 const T2& ID,
                                 const unsigned char& nvXp2,
                                 const unsigned char& nvYp2) {
    X = ID & (((T1)1 << nvXp2) - 1);  // & operation here equals modulo
    Y = (ID >> nvXp2) & (((T1)1 << nvYp2) - 1);
    Z = (ID) >> (nvXp2 + nvYp2);
}

// Packs XYZ components back to a long ID (typically voxelID)
template <typename T1, typename T2>
inline __device__ void IDPacker(T1& ID,
                                const T2& X,
                                const T2& Y,
                                const T2& Z,
                                const unsigned char& nvXp2,
                                const unsigned char& nvYp2) {
    ID = 0;
    ID += X;
    ID += Y << nvXp2;
    ID += Z << (nvXp2 + nvYp2);
}

// From a voxelID to (usually double-precision) xyz coordinate
template <typename T1, typename T2, typename T3>
inline __device__ void voxelID2Position(T1& X,
                                        T1& Y,
                                        T1& Z,
                                        const T2& ID,
                                        const T3& subPosX,
                                        const T3& subPosY,
                                        const T3& subPosZ,
                                        const unsigned char& nvXp2,
                                        const unsigned char& nvYp2,
                                        const T1& voxelSize,
                                        const T1& l) {
    T2 voxelIDX, voxelIDY, voxelIDZ;
    IDChopper<T2, T2>(voxelIDX, voxelIDY, voxelIDZ, ID, nvXp2, nvYp2);
    X = (T1)voxelIDX * voxelSize + (T1)subPosX * l;
    Y = (T1)voxelIDY * voxelSize + (T1)subPosY * l;
    Z = (T1)voxelIDZ * voxelSize + (T1)subPosZ * l;
}

template <typename T1, typename T2>
inline __device__ void applyOriQ2Vector3(T1& X, T1& Y, T1& Z, const T2& Q0, const T2& Q1, const T2& Q2, const T2& Q3) {
    T1 oldX = X;
    T1 oldY = Y;
    T1 oldZ = Z;
    X = ((T2)2.0 * (Q0 * Q0 + Q1 * Q1) - (T2)1.0) * oldX + ((T2)2.0 * (Q1 * Q2 - Q0 * Q3)) * oldY +
        ((T2)2.0 * (Q1 * Q3 + Q0 * Q2)) * oldZ;
    Y = ((T2)2.0 * (Q1 * Q2 + Q0 * Q3)) * oldX + ((T2)2.0 * (Q0 * Q0 + Q2 * Q2) - (T2)1.0) * oldY +
        ((T2)2.0 * (Q2 * Q3 - Q0 * Q1)) * oldZ;
    Z = ((T2)2.0 * (Q1 * Q3 - Q0 * Q2)) * oldX + ((T2)2.0 * (Q2 * Q3 + Q0 * Q1)) * oldY +
        ((T2)2.0 * (Q0 * Q0 + Q3 * Q3) - (T2)1.0) * oldZ;
}

template <typename T1>
inline __device__ T1 distSquared(const T1& x1, const T1& y1, const T1& z1, const T1& x2, const T1& y2, const T1& z2) {
    return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2);
}

// Normalize a 3-component vector
template <typename T1>
inline __device__ void normalizeVector3(T1& x, T1& y, T1& z) {
    T1 magnitude = sqrt(x * x + y * y + z * z);
    // TODO: Think about whether this is safe
    if (magnitude < 1e-6) {
        // printf("Caution!\n");
    }
    x /= magnitude;
    y /= magnitude;
    z /= magnitude;
}

/**
 * Template arguments:
 *   - T1: the floating point accuracy level for contact point location/penetration depth
 *
 * Basic idea: determines whether 2 spheres intersect and the intersection point coordinates
 *
 */
template <typename T1>
inline __device__ void checkSpheresOverlap(const T1& XA,
                                           const T1& YA,
                                           const T1& ZA,
                                           const T1& radA,
                                           const T1& XB,
                                           const T1& YB,
                                           const T1& ZB,
                                           const T1& radB,
                                           T1& CPX,
                                           T1& CPY,
                                           T1& CPZ,
                                           bool& overlap) {
    T1 centerDist2 = distSquared<T1>(XA, YA, ZA, XB, YB, ZB);
    if (centerDist2 > (radA + radB) * (radA + radB)) {
        overlap = false;
        return;
    }
    // If getting this far, then 2 spheres have an intersection, let's calculate the intersection point
    overlap = true;
    T1 A2BVecX = XB - XA;
    T1 A2BVecY = YB - YA;
    T1 A2BVecZ = ZB - ZA;
    normalizeVector3<double>(A2BVecX, A2BVecY, A2BVecZ);
    T1 halfOverlapDepth = (radA + radB - sqrt(centerDist2)) / (T1)2;
    // From center of A, towards center of B, move a distance of radA, then backtrack a bit, for half the overlap depth
    CPX = XA + (radA - halfOverlapDepth) * A2BVecX;
    CPY = YA + (radA - halfOverlapDepth) * A2BVecY;
    CPZ = ZA + (radA - halfOverlapDepth) * A2BVecZ;
}

/**
 * Template arguments:
 *   - T1: the floating point accuracy level for contact point location/penetration depth
 *   - T2: the floating point accuracy level for the relative position of 2 bodies involved
 *
 * Basic idea: this is another version of checkSpheresOverlap which also gives the penetration length and bodyA's
 * outward contact normal
 *
 */
template <typename T1, typename T2>
inline __device__ void checkSpheresOverlap(const T1& XA,
                                           const T1& YA,
                                           const T1& ZA,
                                           const T1& radA,
                                           const T1& XB,
                                           const T1& YB,
                                           const T1& ZB,
                                           const T1& radB,
                                           T1& CPX,
                                           T1& CPY,
                                           T1& CPZ,
                                           T2& normalX,
                                           T2& normalY,
                                           T2& normalZ,
                                           T1& overlapDepth,
                                           bool& overlap) {
    T1 centerDist2 = distSquared<T1>(XA, YA, ZA, XB, YB, ZB);
    if (centerDist2 > (radA + radB) * (radA + radB)) {
        overlap = false;
        return;
    }
    // If getting this far, then 2 spheres have an intersection, let's calculate the intersection point
    overlap = true;
    normalX = XB - XA;
    normalY = YB - YA;
    normalZ = ZB - ZA;
    normalizeVector3<T2>(normalX, normalY, normalZ);
    overlapDepth = radA + radB - sqrt(centerDist2);
    // From center of A, towards center of B, move a distance of radA, then backtrack a bit, for half the overlap depth
    CPX = XA + (radA - overlapDepth / (T1)2) * normalX;
    CPY = YA + (radA - overlapDepth / (T1)2) * normalY;
    CPZ = ZA + (radA - overlapDepth / (T1)2) * normalZ;
}

template <typename T1>
inline __device__ T1
getPointBinID(const double& X, const double& Y, const double& Z, const double& binSize, const T1& nbX, const T1& nbY) {
    T1 binIDX = X / binSize;
    T1 binIDY = Y / binSize;
    T1 binIDZ = Z / binSize;
    return binIDX + binIDY * nbX + binIDZ * nbX * nbY;
}

/**
 * Template arguments:
 *   - T1: the floating point accuracy level for the point coordinate and the frame O coordinate
 *
 * Basic idea: calculate a point's (typically contact point) local coordinate in a specific frame, then return as a
 * float3
 *
 */
template <typename T1>
inline __device__ float3 findLocalCoord(const T1& X,
                                        const T1& Y,
                                        const T1& Z,
                                        const T1& Ox,
                                        const T1& Oy,
                                        const T1& Oz,
                                        const sgps::oriQ_t& oriQ0,
                                        const sgps::oriQ_t& oriQ1,
                                        const sgps::oriQ_t& oriQ2,
                                        const sgps::oriQ_t& oriQ3) {
    float locX, locY, locZ;
    locX = X - Ox;
    locY = Y - Oy;
    locZ = Z - Oz;
    // To find the contact point in the local (body) frame, just apply inverse quaternion to OP vector in global frame
    applyOriQ2Vector3<float, sgps::oriQ_t>(locX, locY, locZ, oriQ0, -oriQ1, -oriQ2, -oriQ3);
    return make_float3(locX, locY, locZ);
}
