#include "hip/hip_runtime.h"
#include <kernel/cudaKernels.cuh>

// *----------------------------------------
// GPU - Testing kernels
__global__ void dynamicTestKernel() {
    printf("Dynamic run\n");
}
__global__ void kinematicTestKernel(sgps::voxelID_default_t* data) {
    if (threadIdx.x == 0) {
        printf("Kinematic run\n");
    }

    if (threadIdx.x < N_INPUT_ITEMS) {
        // data[threadIdx.x] = 2 * data[threadIdx.x] + 1;
        // printf("%d\n", data[threadIdx.x]);
    }
}
// END of GPU Testing kernels
// *----------------------------------------

// *----------------------------------------
// SPH - Kinematic kernels

__global__ void kinematic1stPass(vector3* pos, int n, float tolerance, float radius, int* res_arr) {
    // printf("in kernel\n");
    int idx = threadIdx.x;
    int count = 0;  // count total number of valid contact for the current particle

    if (idx > n) {
        res_arr[idx] = count;
        return;
    }

    for (int i = n - 1; i > idx; i--) {
        float dist2 = (pos[idx].x - pos[i].x) * (pos[idx].x - pos[i].x) +
                      (pos[idx].y - pos[i].y) * (pos[idx].y - pos[i].y) +
                      (pos[idx].z - pos[i].z) * (pos[idx].z - pos[i].z);

        if (dist2 <= (radius * 2 + tolerance) * (radius * 2 + tolerance)) {
            count++;
        }
    }

    res_arr[idx] = count;
}

__global__ void kinematic2ndPass(vector3* pos,
                                 int n,
                                 int* offset,
                                 int* contact_num_arr,
                                 float tolerance,
                                 float radius,
                                 contactData* pair_data) {
    int idx = threadIdx.x;
    if (contact_num_arr[idx] != 0) {
        int cur_idx = offset[idx];

        for (int i = n - 1; i > idx; i--) {
            float dist2 = (pos[idx].x - pos[i].x) * (pos[idx].x - pos[i].x) +
                          (pos[idx].y - pos[i].y) * (pos[idx].y - pos[i].y) +
                          (pos[idx].z - pos[i].z) * (pos[idx].z - pos[i].z);

            if (dist2 < (radius * 2 + tolerance) * (radius * 2 + tolerance)) {
                pair_data[cur_idx].contact_pair.x = idx;
                pair_data[cur_idx].contact_pair.y = i;
                cur_idx++;
            }
        }
    }
    __syncthreads();
}
// END of Kinematic kernels
// *----------------------------------------

// *----------------------------------------
// SPH - Dynamic kernals
__global__ void dynamicPass(contactData* gpu_pair_data,
                            int gpu_pair_n,
                            vector3* gpu_pos,
                            vector3* gpu_vel,
                            vector3* gpu_acc,
                            float radius) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= gpu_pair_n) {
        return;
    }

    float dir_x = gpu_pos[gpu_pair_data[idx].contact_pair.x].x - gpu_pos[gpu_pair_data[idx].contact_pair.y].x;
    float dir_y = gpu_pos[gpu_pair_data[idx].contact_pair.x].y - gpu_pos[gpu_pair_data[idx].contact_pair.y].y;
    float dir_z = gpu_pos[gpu_pair_data[idx].contact_pair.x].z - gpu_pos[gpu_pair_data[idx].contact_pair.y].z;

    float dist2 = dir_x * dir_x + dir_y * dir_y + dir_z * dir_z;
    if (dist2 < radius * radius) {
        // TODO
    }
    __syncthreads();
}
// END of Dynamic kernels
// *----------------------------------------