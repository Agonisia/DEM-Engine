#include "hip/hip_runtime.h"
// Granular force computation related custom kernels
#include <granular/DataStructs.h>

__device__ void applyGravity(unsigned int thisClump, sgps::GranSimParams* simParams, sgps::GranDataDT* granData) {
    // Actually, h should be JITCed into the kernel itself
    granData->h2aX[thisClump] += simParams->h * simParams->h * simParams->Gx;
}
__global__ void deriveClumpForces(sgps::GranSimParams* simParams, sgps::GranDataDT* granData) {
    unsigned int thisClump = blockIdx.x * blockDim.x + threadIdx.x;
    if (thisClump < simParams->nClumpBodies) {
        applyGravity(thisClump, simParams, granData);
    }
}
