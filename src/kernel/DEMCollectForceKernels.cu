#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/DEMDefines.h>
#include <kernel/DEMHelperKernels.cu>

// For analytical entities' owners
__constant__ __device__ smug::bodyID_t objOwner[] = {_objOwner_};
// Mass properties are below, if jitified mass properties are in use
_massDefs_;
_moiDefs_;

__global__ void cashInOwnerIndexA(smug::bodyID_t* idOwner,
                                  smug::bodyID_t* id,
                                  smug::bodyID_t* ownerClumpBody,
                                  smug::contact_t* contactType,
                                  size_t nContactPairs) {
    smug::contactPairs_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        smug::bodyID_t thisBodyID = id[myID];
        idOwner[myID] = ownerClumpBody[thisBodyID];
    }
}

__global__ void cashInOwnerIndexB(smug::bodyID_t* idOwner,
                                  smug::bodyID_t* id,
                                  smug::bodyID_t* ownerClumpBody,
                                  smug::contact_t* contactType,
                                  size_t nContactPairs) {
    smug::contactPairs_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        smug::bodyID_t thisBodyID = id[myID];
        smug::contact_t thisCntType = contactType[myID];
        if (thisCntType == smug::DEM_SPHERE_SPHERE_CONTACT) {
            idOwner[myID] = ownerClumpBody[thisBodyID];
        } else {
            // This is a sphere--analytical geometry contact, its owner is jitified
            idOwner[myID] = objOwner[thisBodyID];
        }
    }
}

/*
__global__ void cashInMassMoiIndex(float* massOwner,
                                   float3* moiOwner,
                                   smug::inertiaOffset_t* inertiaPropOffsets,
                                   smug::bodyID_t* idOwner,
                                   size_t nContactPairs) {
    // _nDistinctMassProperties_  elements are in these arrays
    const float moiX[] = {_moiX_};
    const float moiY[] = {_moiY_};
    const float moiZ[] = {_moiZ_};
    const float MassProperties[] = {_MassProperties_};

    smug::contactPairs_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        smug::bodyID_t thisOwnerID = idOwner[myID];
        smug::inertiaOffset_t myMassOffset = inertiaPropOffsets[thisOwnerID];
        float3 moi;
        moi.x = moiX[myMassOffset];
        moi.y = moiY[myMassOffset];
        moi.z = moiZ[myMassOffset];
        massOwner[myID] = MassProperties[myMassOffset];
        moiOwner[myID] = moi;
    }
}
*/

// computes a ./ b
__global__ void forceToAcc(float3* acc,
                           float3* F,
                           smug::bodyID_t* owner,
                           float modifier,
                           size_t n,
                           smug::DEMDataDT* granData) {
    smug::contactPairs_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < n) {
        float myMass;
        const smug::bodyID_t myOwner = owner[myID];
        // Get my mass info from either jitified arrays or global memory
        // Outputs myMass
        // Use an input named exactly `myOwner' which is the id of this owner
        { _massAcqStrat_; }
        acc[myID] = F[myID] * modifier / myMass;
    }
}

// computes cross(a, b) ./ c
__global__ void forceToAngAcc(float3* angAcc,
                              float3* cntPnt,
                              smug::oriQ_t* oriQw,
                              smug::oriQ_t* oriQx,
                              smug::oriQ_t* oriQy,
                              smug::oriQ_t* oriQz,
                              float3* F,
                              float3* torque_inForceForm,
                              smug::bodyID_t* owner,
                              float modifier,
                              size_t n,
                              smug::DEMDataDT* granData) {
    smug::contactPairs_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < n) {
        const smug::bodyID_t myOwner = owner[myID];
        float3 myMOI;
        // Get my mass info from either jitified arrays or global memory
        // Outputs myMOI
        // Use an input named exactly `myOwner' which is the id of this owner
        { _moiAcqStrat_; }
        const smug::oriQ_t myOriQw = oriQw[myOwner];
        const smug::oriQ_t myOriQx = oriQx[myOwner];
        const smug::oriQ_t myOriQy = oriQy[myOwner];
        const smug::oriQ_t myOriQz = oriQz[myOwner];

        float3 myCntPnt = cntPnt[myID];
        // torque_inForceForm is usually the contribution of rolling resistance and it contributes to torque only, not
        // linear velocity
        float3 myF = (F[myID] + torque_inForceForm[myID]) * modifier;
        // F is in global frame, but it needs to be in local to coordinate with moi and cntPnt
        applyOriQToVector3<float, smug::oriQ_t>(myF.x, myF.y, myF.z, myOriQw, -myOriQx, -myOriQy, -myOriQz);
        angAcc[myID] = cross(myCntPnt, myF) / myMOI;
    }
}

// Place information to an array based on an index array and a value array
__global__ void stashElem(float* out1, float* out2, float* out3, smug::bodyID_t* index, float3* value, size_t n) {
    smug::bodyID_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < n) {
        // my_index is unique, no race condition
        smug::bodyID_t my_index = index[myID];
        float3 my_value = value[myID];
        out1[my_index] += my_value.x;
        out2[my_index] += my_value.y;
        out3[my_index] += my_value.z;
    }
}
