#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
//#include <thirdparty/nvidia_helper_math/helper_math.cuh>
#include <granular/DataStructs.h>
#include <granular/GranularDefines.h>
#include <kernel/DEMHelperKernels.cu>

// Calculate the frictionless force between 2 bodies, return as a float3
// Assumes B2A vector is normalized
inline __device__ float3 calcNormalForce(const double& overlapDepth,
                                         const float3& B2A,
                                         const float3& velB2A,
                                         const float& ARadius,
                                         const float& BRadius,
                                         const float& AOwnerMass,
                                         const float& BOwnerMass,
                                         const sgps::contact_t& contact_type,
                                         const float& E,
                                         const float& CoR) {
    // Note this ad-hoc ``force'' is actually a fake acceleration written in terms of multiples of l
    // float F_mag = 1e18 * overlapDepth;
    // return make_float3(F_mag * B2A.x, F_mag * B2A.y, F_mag * B2A.z);

    // normal component of relative velocity
    const float projection = dot(velB2A, B2A);
    float3 vrel_tan = velB2A - projection * B2A;  // May want to report this for tangent force calculation

    const float mass_eff = (AOwnerMass * BOwnerMass) / (AOwnerMass + BOwnerMass);
    float sqrt_Rd = sqrt(overlapDepth * (ARadius * BRadius) / (ARadius + BRadius));
    const float Sn = 2. * E * sqrt_Rd;

    const float loge = (CoR < SGPS_DEM_TINY_FLOAT) ? log(SGPS_DEM_TINY_FLOAT) : log(CoR);
    float beta = loge / sqrt(loge * loge + SGPS_PI_SQUARED);

    const float k_n = SGPS_TWO_OVER_THREE * Sn;
    const float gamma_n = SGPS_TWO_TIMES_SQRT_FIVE_OVER_SIX * beta * sqrt(Sn * mass_eff);

    // normal force (that A feels)
    // printf("overlapDepth: %f\n", overlapDepth);
    // printf("kn * overlapDepth: %f\n", k_n * overlapDepth);
    // printf("gn * projection: %f\n", gamma_n * projection);
    float3 force = (k_n * overlapDepth + gamma_n * projection) * B2A;
    return force;
}

__global__ void calculateNormalContactForces(sgps::DEMSimParams* simParams,
                                             sgps::DEMDataDT* granData,
                                             size_t nContactPairs,
                                             sgps::DEMTemplate* granTemplates) {
    // CUDA does not support initializing shared arrays, so we have to manually load them
    __shared__ float Radii[_nDistinctClumpComponents_];
    __shared__ float CDRelPosX[_nDistinctClumpComponents_];
    __shared__ float CDRelPosY[_nDistinctClumpComponents_];
    __shared__ float CDRelPosZ[_nDistinctClumpComponents_];
    __shared__ float ClumpMasses[_nDistinctClumpBodyTopologies_];
    if (threadIdx.x < _nActiveLoadingThreads_) {
        const float jitifiedRadii[_nDistinctClumpComponents_] = {_Radii_};
        const float jitifiedCDRelPosX[_nDistinctClumpComponents_] = {_CDRelPosX_};
        const float jitifiedCDRelPosY[_nDistinctClumpComponents_] = {_CDRelPosY_};
        const float jitifiedCDRelPosZ[_nDistinctClumpComponents_] = {_CDRelPosZ_};
        const float jitifiedMass[_nDistinctClumpBodyTopologies_] = {_ClumpMasses_};
        for (sgps::clumpBodyInertiaOffset_t i = threadIdx.x; i < _nDistinctClumpBodyTopologies_;
             i += _nActiveLoadingThreads_) {
            ClumpMasses[i] = jitifiedMass[i];
        }
        for (sgps::clumpComponentOffset_t i = threadIdx.x; i < _nDistinctClumpComponents_;
             i += _nActiveLoadingThreads_) {
            Radii[i] = jitifiedRadii[i];
            CDRelPosX[i] = jitifiedCDRelPosX[i];
            CDRelPosY[i] = jitifiedCDRelPosY[i];
            CDRelPosZ[i] = jitifiedCDRelPosZ[i];
        }
    }
    __syncthreads();

    // First, find relevant bodyIDs, then locate their owners... (how??)

    // But, we will keep everything as is, and test in the end (when cub and jit are in place) how this treatment
    // improves efficiency

    sgps::contactPairs_t myContactID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < nContactPairs) {
        // From a contact ID, grab relevant info on 2 contact bodies
        sgps::bodyID_t bodyA = granData->idGeometryA[myContactID];
        sgps::bodyID_t bodyB = granData->idGeometryB[myContactID];
        sgps::bodyID_t bodyAOwner = granData->ownerClumpBody[bodyA];
        sgps::bodyID_t bodyBOwner = granData->ownerClumpBody[bodyB];
        sgps::clumpComponentOffset_t bodyACompOffset = granData->clumpComponentOffset[bodyA];
        sgps::clumpComponentOffset_t bodyBCompOffset = granData->clumpComponentOffset[bodyB];
        sgps::materialsOffset_t bodyAMatType = granData->materialTupleOffset[bodyA];
        sgps::materialsOffset_t bodyBMatType = granData->materialTupleOffset[bodyB];
        float AOwnerMass = ClumpMasses[granData->inertiaPropOffsets[bodyAOwner]];
        float BOwnerMass = ClumpMasses[granData->inertiaPropOffsets[bodyBOwner]];
        float ARadius = Radii[bodyACompOffset];
        float BRadius = Radii[bodyBCompOffset];

        // Take care of 2 bodies in order, bodyA first, grab location and velocity to local cache
        float3 ALinVel, ARotVel, myRelPos;
        double3 AOwnerPos, bodyAPos;
        sgps::oriQ_t AoriQ0, AoriQ1, AoriQ2, AoriQ3;
        voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
            AOwnerPos.x, AOwnerPos.y, AOwnerPos.z, granData->voxelID[bodyAOwner], granData->locX[bodyAOwner],
            granData->locY[bodyAOwner], granData->locZ[bodyAOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
        myRelPos.x = CDRelPosX[bodyACompOffset];
        myRelPos.y = CDRelPosY[bodyACompOffset];
        myRelPos.z = CDRelPosZ[bodyACompOffset];
        AoriQ0 = granData->oriQ0[bodyAOwner];
        AoriQ1 = granData->oriQ1[bodyAOwner];
        AoriQ2 = granData->oriQ2[bodyAOwner];
        AoriQ3 = granData->oriQ3[bodyAOwner];
        applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPos.x, myRelPos.y, myRelPos.z, AoriQ0, AoriQ1, AoriQ2, AoriQ3);
        bodyAPos.x = AOwnerPos.x + (double)myRelPos.x;
        bodyAPos.y = AOwnerPos.y + (double)myRelPos.y;
        bodyAPos.z = AOwnerPos.z + (double)myRelPos.z;
        ALinVel.x = granData->vX[bodyAOwner];
        ALinVel.y = granData->vY[bodyAOwner];
        ALinVel.z = granData->vZ[bodyAOwner];
        ARotVel.x = granData->omgBarX[bodyAOwner];
        ARotVel.y = granData->omgBarY[bodyAOwner];
        ARotVel.z = granData->omgBarZ[bodyAOwner];

        // Then bodyB, location and velocity
        float3 BLinVel, BRotVel;
        double3 BOwnerPos, bodyBPos;
        sgps::oriQ_t BoriQ0, BoriQ1, BoriQ2, BoriQ3;
        voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
            BOwnerPos.x, BOwnerPos.y, BOwnerPos.z, granData->voxelID[bodyBOwner], granData->locX[bodyBOwner],
            granData->locY[bodyBOwner], granData->locZ[bodyBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
        myRelPos.x = CDRelPosX[bodyBCompOffset];
        myRelPos.y = CDRelPosY[bodyBCompOffset];
        myRelPos.z = CDRelPosZ[bodyBCompOffset];
        BoriQ0 = granData->oriQ0[bodyBOwner];
        BoriQ1 = granData->oriQ1[bodyBOwner];
        BoriQ2 = granData->oriQ2[bodyBOwner];
        BoriQ3 = granData->oriQ3[bodyBOwner];
        applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPos.x, myRelPos.y, myRelPos.z, BoriQ0, BoriQ1, BoriQ2, BoriQ3);
        bodyBPos.x = BOwnerPos.x + (double)myRelPos.x;
        bodyBPos.y = BOwnerPos.y + (double)myRelPos.y;
        bodyBPos.z = BOwnerPos.z + (double)myRelPos.z;
        BLinVel.x = granData->vX[bodyBOwner];
        BLinVel.y = granData->vY[bodyBOwner];
        BLinVel.z = granData->vZ[bodyBOwner];
        BRotVel.x = granData->omgBarX[bodyBOwner];
        BRotVel.y = granData->omgBarY[bodyBOwner];
        BRotVel.z = granData->omgBarZ[bodyBOwner];

        // Now compute the contact point to see if they are truly still in contact
        double3 contactPnt;
        float3 B2A;  // Unit vector pointing from the center of body (sphere) B to body A
        double overlapDepth;
        bool in_contact;
        checkSpheresOverlap<double, float>(bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, bodyBPos.x, bodyBPos.y,
                                           bodyBPos.z, BRadius, contactPnt.x, contactPnt.y, contactPnt.z, B2A.x, B2A.y,
                                           B2A.z, overlapDepth, in_contact);
        if (in_contact) {
            // Instead of add the force to a body-based register, we store it in event-based register, and use CUB to
            // reduce them afterwards
            // atomicAdd(granData->bodyForceX + bodyA, -force * B2AX);
            // atomicAdd(granData->bodyForceX + bodyB, force * B2AX);

            // Right now, only handles contact type 0, sphere--sphere
            sgps::contact_t contact_type = 0;
            // Get k, g, etc. from the (upper-triangle) material property matrix
            unsigned int matEntry = locateMatPair<unsigned int>(bodyAMatType, bodyBMatType);
            float E = granTemplates->EProxy[matEntry];
            // float G = granTemplates->GProxy[matEntry];
            float CoR = granTemplates->CoRProxy[matEntry];

            // Find the contact point in the local (body), but global-axes-aligned frame
            // float3 locCPA = findLocalCoord<double>(contactPntX, contactPntY, contactPntZ, AOwnerX, AOwnerY, AOwnerZ,
            // AoriQ0, AoriQ1, AoriQ2, AoriQ3); float3 locCPB = findLocalCoord<double>(contactPntX, contactPntY,
            // contactPntZ, BOwnerX, BOwnerY, BOwnerZ, BoriQ0, BoriQ1, BoriQ2, BoriQ3);
            float3 locCPA = contactPnt - AOwnerPos;
            float3 locCPB = contactPnt - BOwnerPos;
            // We also need the relative velocity between A and B in global frame to use in the damping terms
            float3 velB2A = (ALinVel + cross(ARotVel, locCPA)) - (BLinVel + cross(BRotVel, locCPB));

            // Calculate contact force
            granData->contactForces[myContactID] = calcNormalForce(overlapDepth, B2A, velB2A, ARadius, BRadius,
                                                                   AOwnerMass, BOwnerMass, contact_type, E, CoR);
            // Write hard-earned results back to global arrays
            granData->contactPointGeometryA[myContactID] = locCPA;
            granData->contactPointGeometryB[myContactID] = locCPB;
        } else {
            granData->contactForces[myContactID] = make_float3(0, 0, 0);
        }
    }
}
