#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
//#include <thirdparty/nvidia_helper_math/helper_math.cuh>
#include <granular/DataStructs.h>
#include <granular/GranularDefines.h>
#include <kernel/DEMHelperKernels.cu>

// Calculate the frictionless force between 2 bodies, return as a float3
// Assumes B2A vector is normalized
inline __device__ float3 calcNormalForce(const double& overlapDepth,
                                         const float3& B2A,
                                         const float3& velB2A,
                                         const float& ARadius,
                                         const float& BRadius,
                                         const float& AOwnerMass,
                                         const float& BOwnerMass,
                                         const sgps::contact_t& contact_type,
                                         const float& E,
                                         const float& CoR) {
    // Note this ad-hoc ``force'' is actually a fake acceleration written in terms of multiples of l
    // float F_mag = 1e18 * overlapDepth;
    // return make_float3(F_mag * B2A.x, F_mag * B2A.y, F_mag * B2A.z);

    // normal component of relative velocity
    const float projection = dot(velB2A, B2A);
    float3 vrel_tan = velB2A - projection * B2A;  // May want to report this for tangent force calculation

    const float mass_eff = (AOwnerMass * BOwnerMass) / (AOwnerMass + BOwnerMass);
    float sqrt_Rd = sqrt(overlapDepth * (ARadius * BRadius) / (ARadius + BRadius));
    const float Sn = 2. * E * sqrt_Rd;

    const float loge = (CoR < SGPS_DEM_TINY_FLOAT) ? log(SGPS_DEM_TINY_FLOAT) : log(CoR);
    float beta = loge / sqrt(loge * loge + SGPS_PI_SQUARED);

    const float k_n = SGPS_TWO_OVER_THREE * Sn;
    const float gamma_n = 2. * SGPS_SQRT_FIVE_OVER_SIX * beta * sqrt(Sn * mass_eff);

    // normal force (that A feels)
    // printf("overlapDepth: %f\n", overlapDepth);
    // printf("kn: %f\n", kn);
    // printf("projection: %f\n", projection);
    // printf("gn: %f\n", gn);
    // printf("Sn: %f\n", Sn);
    // printf("kn * overlapDepth: %f\n", kn * overlapDepth);
    // printf("gn * projection: %f\n", gn * projection);
    float3 force = (k_n * overlapDepth + gamma_n * projection) * B2A;
    return force;
}

__global__ void calculateNormalContactForces(sgps::DEMSimParams* simParams,
                                             sgps::DEMDataDT* granData,
                                             sgps::DEMTemplate* granTemplates) {
    // __shared__ const distinctSphereRadii[@NUM_OF_THAT_ARR@] = {@THAT_ARR@};
    // TODO: These info should be jitfied not brought from global mem
    extern __shared__ float Radii[];
    float* CDRelPosX = Radii + TEST_SHARED_SIZE;
    float* CDRelPosY = Radii + 2 * TEST_SHARED_SIZE;
    float* CDRelPosZ = Radii + 3 * TEST_SHARED_SIZE;
    float* ClumpMasses = Radii + 4 * TEST_SHARED_SIZE;
    if (threadIdx.x == 0) {
        for (unsigned int i = 0; i < simParams->nDistinctClumpComponents; i++) {
            Radii[i] = granTemplates->radiiSphere[i] * simParams->beta;
            CDRelPosX[i] = granTemplates->relPosSphereX[i];
            CDRelPosY[i] = granTemplates->relPosSphereY[i];
            CDRelPosZ[i] = granTemplates->relPosSphereZ[i];
        }
        for (unsigned int i = 0; i < simParams->nDistinctClumpBodyTopologies; i++) {
            ClumpMasses[i] = granTemplates->massClumpBody[i];
        }
    }
    __syncthreads();

    // First, find relevant bodyIDs, then locate their owners... (how??)

    // But, we will keep everything as is, and test in the end (when cub and jit are in place) how this treatment
    // improves efficiency

    sgps::contactPairs_t myContactID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < simParams->nContactPairs) {
        // From a contact ID, grab relevant info on 2 contact bodies
        sgps::bodyID_t bodyA = granData->idGeometryA[myContactID];
        sgps::bodyID_t bodyB = granData->idGeometryB[myContactID];
        sgps::bodyID_t bodyAOwner = granData->ownerClumpBody[bodyA];
        sgps::bodyID_t bodyBOwner = granData->ownerClumpBody[bodyB];
        sgps::clumpComponentOffset_t bodyACompOffset = granData->clumpComponentOffset[bodyA];
        sgps::clumpComponentOffset_t bodyBCompOffset = granData->clumpComponentOffset[bodyB];
        sgps::materialsOffset_t bodyAMatType = granData->materialTupleOffset[bodyA];
        sgps::materialsOffset_t bodyBMatType = granData->materialTupleOffset[bodyB];
        float AOwnerMass = ClumpMasses[granData->inertiaPropOffsets[bodyAOwner]];
        float BOwnerMass = ClumpMasses[granData->inertiaPropOffsets[bodyBOwner]];
        float ARadius = Radii[bodyACompOffset];
        float BRadius = Radii[bodyBCompOffset];

        // Take care of 2 bodies in order, bodyA first, grab location and velocity to local cache
        float3 ALinVel, ARotVel, myRelPos;
        double3 AOwnerPos, bodyAPos;
        sgps::oriQ_t AoriQ0, AoriQ1, AoriQ2, AoriQ3;
        voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
            AOwnerPos.x, AOwnerPos.y, AOwnerPos.z, granData->voxelID[bodyAOwner], granData->locX[bodyAOwner],
            granData->locY[bodyAOwner], granData->locZ[bodyAOwner], simParams->nvXp2, simParams->nvYp2,
            simParams->voxelSize, simParams->l);
        myRelPos.x = CDRelPosX[bodyACompOffset];
        myRelPos.y = CDRelPosY[bodyACompOffset];
        myRelPos.z = CDRelPosZ[bodyACompOffset];
        AoriQ0 = granData->oriQ0[bodyAOwner];
        AoriQ1 = granData->oriQ1[bodyAOwner];
        AoriQ2 = granData->oriQ2[bodyAOwner];
        AoriQ3 = granData->oriQ3[bodyAOwner];
        applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPos.x, myRelPos.y, myRelPos.z, AoriQ0, AoriQ1, AoriQ2, AoriQ3);
        bodyAPos.x = AOwnerPos.x + (double)myRelPos.x;
        bodyAPos.y = AOwnerPos.y + (double)myRelPos.y;
        bodyAPos.z = AOwnerPos.z + (double)myRelPos.z;
        ALinVel.x = granData->hvX[bodyAOwner] * simParams->l / simParams->h;
        ALinVel.y = granData->hvY[bodyAOwner] * simParams->l / simParams->h;
        ALinVel.z = granData->hvZ[bodyAOwner] * simParams->l / simParams->h;
        ARotVel.x = granData->hOmgBarX[bodyAOwner] / simParams->h;
        ARotVel.y = granData->hOmgBarY[bodyAOwner] / simParams->h;
        ARotVel.z = granData->hOmgBarZ[bodyAOwner] / simParams->h;

        // Then bodyB, location and velocity
        float3 BLinVel, BRotVel;
        double3 BOwnerPos, bodyBPos;
        sgps::oriQ_t BoriQ0, BoriQ1, BoriQ2, BoriQ3;
        voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
            BOwnerPos.x, BOwnerPos.y, BOwnerPos.z, granData->voxelID[bodyBOwner], granData->locX[bodyBOwner],
            granData->locY[bodyBOwner], granData->locZ[bodyBOwner], simParams->nvXp2, simParams->nvYp2,
            simParams->voxelSize, simParams->l);
        myRelPos.x = CDRelPosX[bodyBCompOffset];
        myRelPos.y = CDRelPosY[bodyBCompOffset];
        myRelPos.z = CDRelPosZ[bodyBCompOffset];
        BoriQ0 = granData->oriQ0[bodyBOwner];
        BoriQ1 = granData->oriQ1[bodyBOwner];
        BoriQ2 = granData->oriQ2[bodyBOwner];
        BoriQ3 = granData->oriQ3[bodyBOwner];
        applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPos.x, myRelPos.y, myRelPos.z, BoriQ0, BoriQ1, BoriQ2, BoriQ3);
        bodyBPos.x = BOwnerPos.x + (double)myRelPos.x;
        bodyBPos.y = BOwnerPos.y + (double)myRelPos.y;
        bodyBPos.z = BOwnerPos.z + (double)myRelPos.z;
        BLinVel.x = granData->hvX[bodyBOwner] * simParams->l / simParams->h;
        BLinVel.y = granData->hvY[bodyBOwner] * simParams->l / simParams->h;
        BLinVel.z = granData->hvZ[bodyBOwner] * simParams->l / simParams->h;
        BRotVel.x = granData->hOmgBarX[bodyBOwner] / simParams->h;
        BRotVel.y = granData->hOmgBarY[bodyBOwner] / simParams->h;
        BRotVel.z = granData->hOmgBarZ[bodyBOwner] / simParams->h;

        // Now compute the contact point to see if they are truly still in contact
        double3 contactPnt;
        float3 B2A;  // Unit vector pointing from the center of body (sphere) B to body A
        double overlapDepth;
        bool in_contact;
        checkSpheresOverlap<double, float>(bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, bodyBPos.x, bodyBPos.y,
                                           bodyBPos.z, BRadius, contactPnt.x, contactPnt.y, contactPnt.z, B2A.x, B2A.y,
                                           B2A.z, overlapDepth, in_contact);
        if (in_contact) {
            // Instead of add the force to a body-based register, we store it in event-based register, and use CUB to
            // reduce them afterwards
            // atomicAdd(granData->bodyForceX + bodyA, -force * B2AX);
            // atomicAdd(granData->bodyForceX + bodyB, force * B2AX);

            // Right now, only handles contact type 0, sphere--sphere
            sgps::contact_t contact_type = 0;
            // Get k, g, etc. from the (upper-triangle) material property matrix
            unsigned int matEntry = locateMatPair<unsigned int>(bodyAMatType, bodyBMatType);
            float E = granTemplates->EProxy[matEntry];
            // float G = granTemplates->GProxy[matEntry];
            float CoR = granTemplates->CoRProxy[matEntry];

            // Find the contact point in the local (body), but global-axes-aligned frame
            // float3 locCPA = findLocalCoord<double>(contactPntX, contactPntY, contactPntZ, AOwnerX, AOwnerY, AOwnerZ,
            // AoriQ0, AoriQ1, AoriQ2, AoriQ3); float3 locCPB = findLocalCoord<double>(contactPntX, contactPntY,
            // contactPntZ, BOwnerX, BOwnerY, BOwnerZ, BoriQ0, BoriQ1, BoriQ2, BoriQ3);
            float3 locCPA = contactPnt - AOwnerPos;
            float3 locCPB = contactPnt - BOwnerPos;
            // We also need the relative velocity between A and B in global frame to use in the damping terms
            float3 velB2A = (ALinVel + cross(ARotVel, locCPA)) - (BLinVel + cross(BRotVel, locCPB));

            // Calculate contact force
            granData->contactForces[myContactID] = calcNormalForce(overlapDepth, B2A, velB2A, ARadius, BRadius,
                                                                   AOwnerMass, BOwnerMass, contact_type, E, CoR);
            // Write hard-earned results back to global arrays
            granData->contactPointGeometryA[myContactID] = locCPA;
            granData->contactPointGeometryB[myContactID] = locCPB;
        } else {
            granData->contactForces[myContactID] = make_float3(0, 0, 0);
        }
    }
}
