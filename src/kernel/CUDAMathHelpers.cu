#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 *  This file implements common mathematical operations on vector types
 *  (float3, float4 etc.) since these are not provided as standard by CUDA.
 *
 *  The syntax is modeled on the Cg standard library.
 *
 *  This is part of the Helper library includes
 *
 *    Thanks to Linh Hah for additions and fixes.
 */

////////////////////////////////////////////////////////////////////////////////
// Weirdly, a few float3 and double3 operators are not in the cuda toolkit and
// I have difficulty including thridparty cuda helper math here. Even if I can,
// I suspect namespace problems. So they will for now just be defined here manually.
////////////////////////////////////////////////////////////////////////////////

#include <DEM/DEMDefines.h>

inline __device__ float3 cross(float3 a, float3 b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

inline __device__ float dot(float3 a, float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __device__ float length(float3 v) {
    return sqrt(dot(v, v));
}

// Addition and subtraction

inline __device__ float3 operator+(float3 a, float3 b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __device__ float3 operator-(float3 a, float3 b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __device__ float3 operator+(double3 a, double3 b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __device__ float3 operator-(double3 a, double3 b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __device__ void operator+=(float3& a, float3 b) {
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}

inline __device__ void operator-=(float3& a, float3 b) {
    a.x -= b.x;
    a.y -= b.y;
    a.z -= b.z;
}

// Multiplication

inline __device__ float3 operator*(float3 a, float3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
inline __device__ void operator*=(float3& a, float3 b) {
    a.x *= b.x;
    a.y *= b.y;
    a.z *= b.z;
}
inline __device__ float3 operator*(float3 a, float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}
inline __device__ float3 operator*(float b, float3 a) {
    return make_float3(b * a.x, b * a.y, b * a.z);
}
inline __device__ void operator*=(float3& a, float b) {
    a.x *= b;
    a.y *= b;
    a.z *= b;
}

// Division

inline __host__ __device__ float3 operator/(float3 a, float3 b) {
    return ::make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}
inline __host__ __device__ void operator/=(float3& a, float3 b) {
    a.x /= b.x;
    a.y /= b.y;
    a.z /= b.z;
}
inline __host__ __device__ float3 operator/(float3 a, float b) {
    return ::make_float3(a.x / b, a.y / b, a.z / b);
}
inline __host__ __device__ void operator/=(float3& a, float b) {
    a.x /= b;
    a.y /= b;
    a.z /= b;
}

// Cause an error inside a kernel
#define SGPS_DEM_ABORT_KERNEL(...) \
    {                              \
        printf(__VA_ARGS__);       \
        __threadfence();           \
        hipcub::ThreadTrap();         \
    }
