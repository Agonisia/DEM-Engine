#include "hip/hip_runtime.h"
// DEM kernels that does some wildcard stuff, such as modifying the system as per user instruction
#include <kernel/DEMHelperKernels.cu>
#include <DEM/DEMDefines.h>

// Mass properties are below, if jitified mass properties are in use
_massDefs_;
_moiDefs_;

__global__ void applyFamilyChanges(smug::DEMDataDT* granData, size_t nOwnerBodies, float h, float t) {
    smug::bodyID_t myOwner = blockIdx.x * blockDim.x + threadIdx.x;
    if (myOwner < nOwnerBodies) {
        // The user may make references to owner positions, velocities, accelerations and simulation time
        double3 pos;
        float3 vel, acc;
        float mass;
        smug::family_t family_code = granData->familyID[myOwner];
        // Get my mass info from either jitified arrays or global memory
        // Outputs myMass
        // Use an input named exactly `myOwner' which is the id of this owner
        {
            float myMass;
            _massAcqStrat_;
            mass = myMass;
        }
        voxelID2Position<double, smug::voxelID_t, smug::subVoxelPos_t>(
            pos.x, pos.y, pos.z, granData->voxelID[myOwner], granData->locX[myOwner], granData->locY[myOwner],
            granData->locZ[myOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
        vel.x = granData->vX[myOwner];
        vel.y = granData->vY[myOwner];
        vel.z = granData->vZ[myOwner];
        acc.x = granData->aX[myOwner];
        acc.y = granData->aY[myOwner];
        acc.z = granData->aZ[myOwner];

        // Carry out user's instructions
        { _familyChangeRules_; }
    }
}
