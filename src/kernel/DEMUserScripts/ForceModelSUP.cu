#include "hip/hip_runtime.h"
// Based on FullHertzianForceModel.cu, modified for Scaled-Up Particle(SUP) model
// Reference: "Inter-particle torque scaling in coarse grained DEM with rolling resistance 
// and particle size distributions" - Hu et al., Powder Technology 438 (2024)

// Acquire scale_factor_l
float l = scale_factor_l;
// Ensure l is valid, default to 1.0 if not properly set or <= 0.
if (l < 1e-5f) { // Using a small epsilon
    l = 1.0f;
}

float overlap_s = overlapDepth;
// If no contact, clear history and exit
if (overlap_s > 0) {
    // ========================================================================
    // SUP Step 1: Input Scaling (Convert Scaled-up variables to Original variables)
    // Based on Table 1 from the paper
    // ========================================================================
    
    // Overlap scaling: δ_O = δ_S / l
    float overlap_o = overlap_s / l;

    if (overlap_o > 0.f) {
        // Material properties from the original model structure
        float E_cnt, G_cnt, CoR_cnt, mu_cnt, Crr_cnt;
        {
            float E_A_orig = E[bodyAMatType]; 
            float nu_A_orig = nu[bodyAMatType];
            float E_B_orig = E[bodyBMatType];
            float nu_B_orig = nu[bodyBMatType];
            matProxy2ContactParam<float>(E_cnt, G_cnt, E_A_orig, nu_A_orig, E_B_orig, nu_B_orig);
            CoR_cnt = CoR[bodyAMatType][bodyBMatType];
            mu_cnt = mu[bodyAMatType][bodyBMatType];
            Crr_cnt = Crr[bodyAMatType][bodyBMatType];
        }

        // Radius scaling: R_O = R_S / l
        float R_s_A = ARadius;
        float R_s_B = BRadius;
        float R_o_A = R_s_A / l;
        float R_o_B = R_s_B / l;

        // Contact point scaling: position_O = position_S / l
        float3 locCPA_s = locCPA;
        float3 locCPB_s = locCPB;
        float3 locCPA_o = locCPA_s / l;
        float3 locCPB_o = locCPB_s / l;

        // Rotational velocity scaling: ω_O = ω_S / l (CORRECTED from ω_S * l)
        // According to Table 1: ω_O = l·ω_S means ω_O = ω_S / l in the original scale
        float3 ARotVel_s = ARotVel;
        float3 BRotVel_s = BRotVel;
        float3 ARotVel_o = ARotVel_s / l;  // CORRECTED
        float3 BRotVel_o = BRotVel_s / l;  // CORRECTED

        // Calculate rotational velocities at contact points in original scale
        float3 rotVelCPA_o_local = cross(ARotVel_o, locCPA_o);
        float3 rotVelCPB_o_local = cross(BRotVel_o, locCPB_o);
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPA_o_local.x, rotVelCPA_o_local.y, rotVelCPA_o_local.z, AOriQ.w, AOriQ.x, AOriQ.y, AOriQ.z);
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPB_o_local.x, rotVelCPB_o_local.y, rotVelCPB_o_local.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);

        // Mass scaling: m_O = m_S / l³
        float mass_s_A = AOwnerMass;
        float mass_s_B = BOwnerMass;
        float mass_o_A = mass_s_A / (l * l * l);
        float mass_o_B = mass_s_B / (l * l * l);

        if (mass_o_A <= 0.f) {
            mass_o_A = 1e-12f
        };
        if (mass_o_B <= 0.f) {
            mass_o_B = 1e-12f
        };

        // Time step scaling: Δt_O = Δt_S / l
        float ts_s = ts;
        float ts_o = ts_s / l;

        // Tangential displacement history scaling
        float3 delta_tan_s = make_float3(delta_tan_x, delta_tan_y, delta_tan_z);
        float3 delta_tan_o = delta_tan_s / l;

        float delta_time_s = delta_time;
        float delta_time_o = delta_time_s / l;

        // Variables for force calculation at original scale
        float mass_eff_o, sqrt_Rd_o, beta_o;
        float R_star_o;
        float3 vrel_tan_o;

        // Initialize force components at original scale
        float3 F_normal_o_vec = make_float3(0.f, 0.f, 0.f);
        float3 F_tangential_o_vec = make_float3(0.f, 0.f, 0.f);
        float3 torque_only_force_o = make_float3(0.f, 0.f, 0.f);

        // ========================================================================
        // SUP Step 2: Calculate forces at original scale
        // ========================================================================
        {
            // Calculate relative velocity at original scale
            const float3 velB2A_o = (ALinVel + rotVelCPA_o_local) - (BLinVel + rotVelCPB_o_local);
            const float projection_o = dot(velB2A_o, B2A);
            vrel_tan_o = velB2A_o - projection_o * B2A;

            // Update tangential displacement history
            {
                delta_tan_o += ts_o * vrel_tan_o;
                const float disp_proj_o = dot(delta_tan_o, B2A);
                delta_tan_o -= disp_proj_o * B2A;
                delta_time_o += ts_o;
            }

            // Calculate effective mass
            mass_eff_o = (mass_o_A * mass_o_B) / (mass_o_A + mass_o_B);
            if (mass_o_A <= 1e-12f && mass_o_B <= 1e-12f) {
                mass_eff_o = 1e-6f;
            } else if (mass_o_A <= 1e-12f) {
                mass_eff_o = mass_o_B;
            } else if (mass_o_B <= 1e-12f) {
                mass_eff_o = mass_o_A;
            }

            // Calculate effective radius
            if (R_o_A <= 1.0e-12f && R_o_B <= 1.0e-12f) { R_star_o = 1.0e-6f; }
            else if (R_o_A <= 1.0e-12f) { R_star_o = R_o_B; }
            else if (R_o_B <= 1.0e-12f) { R_star_o = R_o_A; }
            else { R_star_o = (R_o_A * R_o_B) / (R_o_A + R_o_B); }

            // Calculate normal force using Hertzian contact model
            sqrt_Rd_o = sqrtf(overlap_o * R_star_o);
            const float Sn_o = 2.f * E_cnt * sqrt_Rd_o;

            const float loge_o = (CoR_cnt < DEME_TINY_FLOAT) ? logf(DEME_TINY_FLOAT) : logf(CoR_cnt);
            beta_o = loge_o / sqrtf(loge_o * loge_o + deme::PI_SQUARED);

            const float k_n_o = deme::TWO_OVER_THREE * Sn_o;
            const float gamma_n_o = deme::TWO_TIMES_SQRT_FIVE_OVER_SIX * beta_o * sqrtf(Sn_o * mass_eff_o);

            F_normal_o_vec = (k_n_o * overlap_o + gamma_n_o * projection_o) * B2A;
        }

        // Calculate rolling resistance if enabled
        if (Crr_cnt > 0.0f) {
            bool should_add_rolling_resistance_o = true;
            {
                const float R_eff_o = R_star_o;
                const float kn_simple_o = deme::FOUR_OVER_THREE * E_cnt * sqrtf(R_eff_o);
                const float gn_simple_o = -2.f * sqrtf(deme::FIVE_OVER_THREE * mass_eff_o * E_cnt) * beta_o * powf(R_eff_o, 0.25f);
                const float d_coeff_o = gn_simple_o / (2.f * sqrtf(kn_simple_o * mass_eff_o));

                if (d_coeff_o < 1.0f) {
                    float t_collision_o = deme::PI * sqrtf(mass_eff_o / (kn_simple_o * (1.f - d_coeff_o * d_coeff_o)));
                    if (delta_time_o <= t_collision_o) {
                        should_add_rolling_resistance_o = false;
                    }
                }
            }
            if (should_add_rolling_resistance_o) {
                const float3 v_rot_o_global = rotVelCPB_o_local - rotVelCPA_o_local;
                const float v_rot_o_mag = length(v_rot_o_global);
                if (v_rot_o_mag > DEME_TINY_FLOAT) {
                    torque_only_force_o = (v_rot_o_global / v_rot_o_mag) * (Crr_cnt * length(F_normal_o_vec));
                }
            }
        }

        // Calculate tangential force if friction is enabled
        if (mu_cnt > 0.0f) {
            const float kt_o = 8.f * G_cnt * sqrt_Rd_o;
            const float gt_o = -deme::TWO_TIMES_SQRT_FIVE_OVER_SIX * beta_o * sqrtf(mass_eff_o * kt_o);
            float3 tangent_force_trial_o = -kt_o * delta_tan_o - gt_o * vrel_tan_o;

            const float ft_o_mag_trial = length(tangent_force_trial_o);
            if (ft_o_mag_trial > DEME_TINY_FLOAT) {
                const float ft_max_o = length(F_normal_o_vec) * mu_cnt;
                if (ft_o_mag_trial > ft_max_o) {
                    // Sliding friction
                    F_tangential_o_vec = (ft_max_o / ft_o_mag_trial) * tangent_force_trial_o;
                    if (fabs(kt_o) > DEME_TINY_FLOAT) {
                        delta_tan_o = (F_tangential_o_vec + gt_o * vrel_tan_o) / (-kt_o);
                    } else {
                        delta_tan_o = make_float3(0.f,0.f,0.f);
                    }
                } else {
                    // Static friction
                    F_tangential_o_vec = tangent_force_trial_o;
                }
            } else {
                F_tangential_o_vec = make_float3(0.f, 0.f, 0.f);
            }
        }

        // ========================================================================
        // SUP Step 3: Scale forces back to scaled-up system
        // According to Eq. (23): F_IS = l²·F_IO
        // ========================================================================
        float l_sq = l * l;
        float3 F_total_o_vec = F_normal_o_vec + F_tangential_o_vec + torque_only_force_o;
        force = F_total_o_vec * l_sq;

        // Note: If torques are needed separately for rotation calculation,
        // they should also be scaled by l² according to Eq. (25): M_IS = l²·M_IO
        // Example (if needed):
        // float3 torque_o = cross(locCPA_o, F_total_o_vec);
        // torque = torque_o * l_sq;

        // ========================================================================
        // SUP Step 4: Update history variables in scaled-up system
        // ========================================================================
        delta_tan_s = delta_tan_o * l;
        delta_tan_x = delta_tan_s.x;
        delta_tan_y = delta_tan_s.y;
        delta_tan_z = delta_tan_s.z;

        delta_time_s = delta_time_o * l;
        delta_time = delta_time_s;
    } else { 
        // No contact at original scale
        delta_time = 0;
        delta_tan_x = 0;
        delta_tan_y = 0;
        delta_tan_z = 0;
    }
} else {
    // No contact at scaled-up scale
    delta_time = 0;
    delta_tan_x = 0;
    delta_tan_y = 0;
    delta_tan_z = 0;
}