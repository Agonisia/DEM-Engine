#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/DEMDefines.h>
#include <kernel/DEMHelperKernels.cu>

inline __device__ void cleanUpContactForces(size_t thisContact,
                                            sgps::DEMSimParams* simParams,
                                            sgps::DEMDataDT* granData) {
    granData->contactForces[thisContact].x = 0;
    granData->contactForces[thisContact].y = 0;
    granData->contactForces[thisContact].z = 0;
}

inline __device__ void cleanUpAcc(size_t thisClump, sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData) {
    granData->aX[thisClump] = _Gx_;
    granData->aY[thisClump] = _Gy_;
    granData->aZ[thisClump] = _Gz_;
    granData->alphaX[thisClump] = 0;
    granData->alphaY[thisClump] = 0;
    granData->alphaZ[thisClump] = 0;
}

__global__ void prepareForceArrays(sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData, size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        cleanUpContactForces(myID, simParams, granData);
    }
    if (myID < _nOwnerBodies_) {
        cleanUpAcc(myID, simParams, granData);
    }
}