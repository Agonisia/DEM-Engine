#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/DEMDefines.h>
#include <kernel/DEMHelperKernels.cu>

inline __device__ void cleanUpContactForces(size_t thisContact,
                                            sgps::DEMSimParams* simParams,
                                            sgps::DEMDataDT* granData) {
    const float3 zeros = make_float3(0, 0, 0);
    granData->contactForces[thisContact] = zeros;
    granData->contactTorque_convToForce[thisContact] = zeros;
    // TODO: Prescribed forces to be added here
}

inline __device__ void cleanUpAcc(size_t thisClump, sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData) {
    granData->aX[thisClump] = 0;
    granData->aY[thisClump] = 0;
    granData->aZ[thisClump] = 0;
    granData->alphaX[thisClump] = 0;
    granData->alphaY[thisClump] = 0;
    granData->alphaZ[thisClump] = 0;
    // TODO: Prescribed accelerations to be added here
}

__global__ void prepareForceArrays(sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData, size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        cleanUpContactForces(myID, simParams, granData);
    }
    if (myID < simParams->nOwnerBodies) {
        cleanUpAcc(myID, simParams, granData);
    }
}

__global__ void rearrangeContactHistory(sgps::contactPairs_t* contactMapping,
                                        float3* oldHistory,
                                        float* oldDuration,
                                        float3* newHistory,
                                        float* newDuration,
                                        sgps::notStupidBool_t* sentry,
                                        size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        sgps::contactPairs_t map_from = contactMapping[myID];
        if (map_from == sgps::DEM_NULL_MAPPING_PARTNER) {
            // If it is a NULL ID then kT says this contact is new
            newHistory[myID] = make_float3(0, 0, 0);
            newDuration[myID] = 0;
        } else {
            // Not a new contact, need to map it from somewhere in the old history array
            newHistory[myID] = oldHistory[map_from];
            newDuration[myID] = oldDuration[map_from];
            // This sentry trys to make sure that all `alive' contacts got mapped to some place
            sentry[map_from] = 0;
        }
    }
}

__global__ void markAliveContacts(float* duration, sgps::notStupidBool_t* sentry, size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        float myDuration = duration[myID];
        // If this is alive then mark it
        if (myDuration > SGPS_DEM_TINY_FLOAT) {
            sentry[myID] = 1;
        } else {
            sentry[myID] = 0;
        }
    }
}
