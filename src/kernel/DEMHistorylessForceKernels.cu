#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
//#include <thirdparty/nvidia_helper_math/helper_math.cuh>
#include <DEM/DEMDefines.h>
#include <kernel/DEMHelperKernels.cu>

__global__ void calculateContactForces(sgps::DEMSimParams* simParams, sgps::DEMDataDT* granData, size_t nContactPairs) {
    // _nDistinctMassProperties_ or _nJitifiableClumpComponents_ elements are in these arrays
    const float Radii[] = {_Radii_};
    const float CDRelPosX[] = {_CDRelPosX_};
    const float CDRelPosY[] = {_CDRelPosY_};
    const float CDRelPosZ[] = {_CDRelPosZ_};
    const float MassProperties[] = {_MassProperties_};

    // _nMatTuples_ elements are in these arrays
    const float EProxy[] = {_EProxy_};
    const float nuProxy[] = {_nuProxy_};
    const float CoRProxy[] = {_CoRProxy_};

    // _nAnalGM_ elements are in these arrays
    const sgps::objType_t objType[_nAnalGMSafe_] = {_objType_};
    const sgps::bodyID_t objOwner[_nAnalGMSafe_] = {_objOwner_};
    const bool objNormal[_nAnalGMSafe_] = {_objNormal_};
    const sgps::materialsOffset_t objMaterial[_nAnalGMSafe_] = {_objMaterial_};
    const float objRelPosX[_nAnalGMSafe_] = {_objRelPosX_};
    const float objRelPosY[_nAnalGMSafe_] = {_objRelPosY_};
    const float objRelPosZ[_nAnalGMSafe_] = {_objRelPosZ_};
    const float objRotX[_nAnalGMSafe_] = {_objRotX_};
    const float objRotY[_nAnalGMSafe_] = {_objRotY_};
    const float objRotZ[_nAnalGMSafe_] = {_objRotZ_};
    const float objSize1[_nAnalGMSafe_] = {_objSize1_};
    const float objSize2[_nAnalGMSafe_] = {_objSize2_};
    const float objSize3[_nAnalGMSafe_] = {_objSize3_};

    sgps::contactPairs_t myContactID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < nContactPairs) {
        // Identify contact type first
        sgps::contact_t myContactType = granData->contactType[myContactID];
        // Allocate the registers needed
        double3 contactPnt;
        float3 B2A;  // Unit vector pointing from body B to body A
        double overlapDepth;
        double3 AOwnerPos, bodyAPos, BOwnerPos, bodyBPos;
        float3 ALinVel, ARotVel, rotVelCPA, BLinVel, BRotVel, rotVelCPB;
        float AOwnerMass, ARadius, BOwnerMass, BRadius;
        sgps::materialsOffset_t bodyAMatType, bodyBMatType;
        sgps::oriQ_t AoriQ0, AoriQ1, AoriQ2, AoriQ3;
        sgps::oriQ_t BoriQ0, BoriQ1, BoriQ2, BoriQ3;
        sgps::family_t AOwnerFamily, BOwnerFamily;
        // Take care of 2 bodies in order, bodyA first, grab location and velocity to local cache
        // We know in this kernel, bodyA will be a sphere; bodyB can be something else
        {
            sgps::bodyID_t sphereID = granData->idGeometryA[myContactID];
            sgps::bodyID_t bodyAOwner = granData->ownerClumpBody[sphereID];

            float myRelPosX, myRelPosY, myRelPosZ, myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPosXYZ, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            bodyAMatType = granData->materialTupleOffset[sphereID];
            AOwnerMass = MassProperties[granData->inertiaPropOffsets[bodyAOwner]];

            AOwnerFamily = granData->familyID[bodyAOwner];
            voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
                AOwnerPos.x, AOwnerPos.y, AOwnerPos.z, granData->voxelID[bodyAOwner], granData->locX[bodyAOwner],
                granData->locY[bodyAOwner], granData->locZ[bodyAOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            AoriQ0 = granData->oriQ0[bodyAOwner];
            AoriQ1 = granData->oriQ1[bodyAOwner];
            AoriQ2 = granData->oriQ2[bodyAOwner];
            AoriQ3 = granData->oriQ3[bodyAOwner];
            applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPosX, myRelPosY, myRelPosZ, AoriQ0, AoriQ1, AoriQ2, AoriQ3);
            bodyAPos.x = AOwnerPos.x + (double)myRelPosX;
            bodyAPos.y = AOwnerPos.y + (double)myRelPosY;
            bodyAPos.z = AOwnerPos.z + (double)myRelPosZ;
            ALinVel.x = granData->vX[bodyAOwner];
            ALinVel.y = granData->vY[bodyAOwner];
            ALinVel.z = granData->vZ[bodyAOwner];
            ARotVel.x = granData->omgBarX[bodyAOwner];
            ARotVel.y = granData->omgBarY[bodyAOwner];
            ARotVel.z = granData->omgBarZ[bodyAOwner];
            ARadius = myRadius;
        }

        // Then bodyB, location and velocity
        if (myContactType == sgps::DEM_SPHERE_SPHERE_CONTACT) {
            sgps::bodyID_t sphereID = granData->idGeometryB[myContactID];
            sgps::bodyID_t bodyBOwner = granData->ownerClumpBody[sphereID];

            float myRelPosX, myRelPosY, myRelPosZ, myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPosXYZ, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            bodyBMatType = granData->materialTupleOffset[sphereID];
            BOwnerMass = MassProperties[granData->inertiaPropOffsets[bodyBOwner]];

            BOwnerFamily = granData->familyID[bodyBOwner];
            voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
                BOwnerPos.x, BOwnerPos.y, BOwnerPos.z, granData->voxelID[bodyBOwner], granData->locX[bodyBOwner],
                granData->locY[bodyBOwner], granData->locZ[bodyBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            BoriQ0 = granData->oriQ0[bodyBOwner];
            BoriQ1 = granData->oriQ1[bodyBOwner];
            BoriQ2 = granData->oriQ2[bodyBOwner];
            BoriQ3 = granData->oriQ3[bodyBOwner];
            applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPosX, myRelPosY, myRelPosZ, BoriQ0, BoriQ1, BoriQ2, BoriQ3);
            bodyBPos.x = BOwnerPos.x + (double)myRelPosX;
            bodyBPos.y = BOwnerPos.y + (double)myRelPosY;
            bodyBPos.z = BOwnerPos.z + (double)myRelPosZ;
            BLinVel.x = granData->vX[bodyBOwner];
            BLinVel.y = granData->vY[bodyBOwner];
            BLinVel.z = granData->vZ[bodyBOwner];
            BRotVel.x = granData->omgBarX[bodyBOwner];
            BRotVel.y = granData->omgBarY[bodyBOwner];
            BRotVel.z = granData->omgBarZ[bodyBOwner];
            BRadius = myRadius;

            myContactType = checkSpheresOverlap<double, float>(
                bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, bodyBPos.x, bodyBPos.y, bodyBPos.z, BRadius, contactPnt.x,
                contactPnt.y, contactPnt.z, B2A.x, B2A.y, B2A.z, overlapDepth);
        } else {
            // If B is analytical entity, its owner, relative location, material info is jitified
            sgps::objID_t bodyB = granData->idGeometryB[myContactID];
            sgps::bodyID_t bodyBOwner = objOwner[bodyB];
            bodyBMatType = objMaterial[bodyB];
            BOwnerMass = MassProperties[granData->inertiaPropOffsets[bodyBOwner]];
            // TODO: fix these...
            BRadius = 10000.f;
            float myRelPosX, myRelPosY, myRelPosZ;
            float3 bodyBRot;

            voxelID2Position<double, sgps::voxelID_t, sgps::subVoxelPos_t>(
                BOwnerPos.x, BOwnerPos.y, BOwnerPos.z, granData->voxelID[bodyBOwner], granData->locX[bodyBOwner],
                granData->locY[bodyBOwner], granData->locZ[bodyBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            myRelPosX = objRelPosX[bodyB];
            myRelPosY = objRelPosY[bodyB];
            myRelPosZ = objRelPosZ[bodyB];
            BoriQ0 = granData->oriQ0[bodyBOwner];
            BoriQ1 = granData->oriQ1[bodyBOwner];
            BoriQ2 = granData->oriQ2[bodyBOwner];
            BoriQ3 = granData->oriQ3[bodyBOwner];
            applyOriQ2Vector3<float, sgps::oriQ_t>(myRelPosX, myRelPosY, myRelPosZ, BoriQ0, BoriQ1, BoriQ2, BoriQ3);
            bodyBPos.x = BOwnerPos.x + (double)myRelPosX;
            bodyBPos.y = BOwnerPos.y + (double)myRelPosY;
            bodyBPos.z = BOwnerPos.z + (double)myRelPosZ;

            // B's orientation (such as plane normal) is rotated with its owner too
            bodyBRot.x = objRotX[bodyB];
            bodyBRot.y = objRotY[bodyB];
            bodyBRot.z = objRotZ[bodyB];
            applyOriQ2Vector3<float, sgps::oriQ_t>(bodyBRot.x, bodyBRot.y, bodyBRot.z, BoriQ0, BoriQ1, BoriQ2, BoriQ3);

            BLinVel.x = granData->vX[bodyBOwner];
            BLinVel.y = granData->vY[bodyBOwner];
            BLinVel.z = granData->vZ[bodyBOwner];
            BRotVel.x = granData->omgBarX[bodyBOwner];
            BRotVel.y = granData->omgBarY[bodyBOwner];
            BRotVel.z = granData->omgBarZ[bodyBOwner];

            // Note for this test on dT side we don't enlarge entities
            myContactType = checkSphereEntityOverlap<double>(
                bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, objType[bodyB], bodyBPos.x, bodyBPos.y, bodyBPos.z,
                bodyBRot.x, bodyBRot.y, bodyBRot.z, objSize1[bodyB], objSize2[bodyB], objSize3[bodyB], objNormal[bodyB],
                0.0, contactPnt.x, contactPnt.y, contactPnt.z, B2A.x, B2A.y, B2A.z, overlapDepth);
        }

        if (myContactType != sgps::DEM_NOT_A_CONTACT) {
            // Material properties and time (user referrable)
            float E, CoR, h;
            {
                h = simParams->h;
                matProxy2ContactParam<float>(E, CoR, EProxy[bodyAMatType], nuProxy[bodyAMatType],
                                             CoRProxy[bodyAMatType], EProxy[bodyBMatType], nuProxy[bodyBMatType],
                                             CoRProxy[bodyBMatType]);
            }
            // Variables that we need to report back (user referrable)
            float3 force = make_float3(0, 0, 0);
            {
                float3 locCPA = contactPnt - AOwnerPos;
                float3 locCPB = contactPnt - BOwnerPos;
                // Now map this contact point location to bodies' local ref
                applyOriQ2Vector3<float, sgps::oriQ_t>(locCPA.x, locCPA.y, locCPA.z, AoriQ0, -AoriQ1, -AoriQ2, -AoriQ3);
                applyOriQ2Vector3<float, sgps::oriQ_t>(locCPB.x, locCPB.y, locCPB.z, BoriQ0, -BoriQ1, -BoriQ2, -BoriQ3);
                granData->contactPointGeometryA[myContactID] = locCPA;
                granData->contactPointGeometryB[myContactID] = locCPB;
                // We also need the relative velocity between A and B in global frame to use in the damping terms
                // To get that, we need contact points' rotational velocity in GLOBAL frame
                rotVelCPA = cross(ARotVel, locCPA);
                rotVelCPB = cross(BRotVel, locCPB);
                applyOriQ2Vector3<float, sgps::oriQ_t>(rotVelCPA.x, rotVelCPA.y, rotVelCPA.z, AoriQ0, AoriQ1, AoriQ2,
                                                       AoriQ3);
                applyOriQ2Vector3<float, sgps::oriQ_t>(rotVelCPB.x, rotVelCPB.y, rotVelCPB.z, BoriQ0, BoriQ1, BoriQ2,
                                                       BoriQ3);
            }

            // The following part, the force model, is user-specifiable
            // NOTE!! "force" must be properly set by this piece of code
            { _DEMForceModel_; }

            // Write hard-earned values back to global memory
            granData->contactForces[myContactID] = force;
        } else {
            granData->contactForces[myContactID] = make_float3(0, 0, 0);
        }
    }
}
