#include "hip/hip_runtime.h"
#include <sph/datastruct.h>

// *----------------------------------------
// SPH - Kinematic kernels

__global__ void kinematic1stPass(vector3* pos, int n, float tolerance, float radius, int* res_arr) {
    // printf("in kernel\n");
    int idx = threadIdx.x;
    int count = 0;  // count total number of valid contact for the current particle

    if (idx > n) {
        res_arr[idx] = count;
        return;
    }

    for (int i = n - 1; i > idx; i--) {
        float dist2 = (pos[idx].x - pos[i].x) * (pos[idx].x - pos[i].x) +
                      (pos[idx].y - pos[i].y) * (pos[idx].y - pos[i].y) +
                      (pos[idx].z - pos[i].z) * (pos[idx].z - pos[i].z);

        if (dist2 <= (radius * 2 + tolerance) * (radius * 2 + tolerance)) {
            count++;
        }
    }

    res_arr[idx] = count;
}

__global__ void kinematic2ndPass(vector3* pos,
                                 int n,
                                 int* offset,
                                 int* contact_num_arr,
                                 float tolerance,
                                 float radius,
                                 contactData* pair_data) {
    int idx = threadIdx.x;

    if (contact_num_arr[idx] != 0) {
        int cur_idx = offset[idx];

        for (int i = n - 1; i > idx; i--) {
            float dist2 = (pos[idx].x - pos[i].x) * (pos[idx].x - pos[i].x) +
                          (pos[idx].y - pos[i].y) * (pos[idx].y - pos[i].y) +
                          (pos[idx].z - pos[i].z) * (pos[idx].z - pos[i].z);

            if (dist2 < ((radius * 2 + tolerance) * (radius * 2 + tolerance))) {
                pair_data[cur_idx].contact_pair.x = idx;
                pair_data[cur_idx].contact_pair.y = i;
                cur_idx++;
            }
        }
    }
    __syncthreads();
}
// END of Kinematic kernels
// *----------------------------------------

// *----------------------------------------
// SPH - Dynamic kernals
__global__ void dynamic1stPass(contactData* gpu_pair_data,
                               int gpu_pair_n,
                               vector3* gpu_pos,
                               vector3* gpu_vel,
                               vector3* gpu_acc,
                               bool* gpu_fix,
                               float radius) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= gpu_pair_n) {
        return;
    }

    float dir_x = gpu_pos[gpu_pair_data[idx].contact_pair.x].x - gpu_pos[gpu_pair_data[idx].contact_pair.y].x;
    float dir_y = gpu_pos[gpu_pair_data[idx].contact_pair.x].y - gpu_pos[gpu_pair_data[idx].contact_pair.y].y;
    float dir_z = gpu_pos[gpu_pair_data[idx].contact_pair.x].z - gpu_pos[gpu_pair_data[idx].contact_pair.y].z;

    float dist2 = dir_x * dir_x + dir_y * dir_y + dir_z * dir_z;

    if (dist2 < (2 * radius) * (2 * radius)) {
        float coe = 1000.f;

        if (gpu_fix[gpu_pair_data[idx].contact_pair.x] == false) {
            gpu_acc[gpu_pair_data[idx].contact_pair.x].x = dir_x * coe;
            gpu_acc[gpu_pair_data[idx].contact_pair.x].y = dir_y * coe;
            gpu_acc[gpu_pair_data[idx].contact_pair.x].z = dir_z * coe;
        }

        if (gpu_fix[gpu_pair_data[idx].contact_pair.y] == false) {
            gpu_acc[gpu_pair_data[idx].contact_pair.y].x = -dir_x * coe;
            gpu_acc[gpu_pair_data[idx].contact_pair.y].y = -dir_y * coe;
            gpu_acc[gpu_pair_data[idx].contact_pair.y].z = -dir_z * coe;
        }
    }

    __syncthreads();
}

__global__ void dynamic2ndPass(vector3* gpu_pos,
                               vector3* gpu_vel,
                               vector3* gpu_acc,
                               bool* gpu_fix,
                               int gpu_n,
                               float time_step,
                               float radius) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= gpu_n) {
        return;
    }

    if (gpu_fix[idx] == false) {
        float grav = -9.8f;
        gpu_acc[idx].z = gpu_acc[idx].z + grav;
    }

    gpu_vel[idx].x = gpu_vel[idx].x + gpu_acc[idx].x * time_step;
    gpu_vel[idx].y = gpu_vel[idx].y + gpu_acc[idx].y * time_step;
    gpu_vel[idx].z = gpu_vel[idx].z + gpu_acc[idx].z * time_step;

    gpu_pos[idx].x = gpu_pos[idx].x + gpu_vel[idx].x * time_step;
    gpu_pos[idx].y = gpu_pos[idx].y + gpu_vel[idx].y * time_step;
    gpu_pos[idx].z = gpu_pos[idx].z + gpu_vel[idx].z * time_step;

    gpu_acc[idx].x = 0.f;
    gpu_acc[idx].y = 0.f;
    gpu_acc[idx].z = 0.f;
}

__global__ void testKernel() {
    printf("test run\n");
}
// END of Dynamic kernels
// *----------------------------------------