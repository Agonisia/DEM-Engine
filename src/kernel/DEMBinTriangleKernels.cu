#include "hip/hip_runtime.h"
// DEM bin--sphere relations-related custom kernels
#include <DEM/Defines.h>
#include <kernel/DEMHelperKernels.cu>

__global__ void getNumberOfBinsEachTriangleTouches(deme::DEMSimParams* simParams,
                                                   deme::DEMDataKT* granData,
                                                   deme::binsTriangleTouches_t* numBinsTriTouches) {
    deme::triID_t triID = blockIdx.x * blockDim.x + threadIdx.x;
    if (triID < simParams->nTriGM) {
        // Sphere's family ID
        unsigned int triFamilyNum;
        double myPosX, myPosY, myPosZ;
        double myRadius;
        {
            // My sphere voxel ID and my relPos
            deme::bodyID_t myOwnerID = granData->ownerMesh[triID];
            triFamilyNum = granData->familyID[myOwnerID];
            float myRelPosX, myRelPosY, myRelPosZ;
            double ownerX, ownerY, ownerZ;

            // Get my component offset info from global array
            granData->relPosNode1[triID];

            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[myOwnerID], granData->locX[myOwnerID],
                granData->locY[myOwnerID], granData->locZ[myOwnerID], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            const float myOriQw = granData->oriQw[myOwnerID];
            const float myOriQx = granData->oriQx[myOwnerID];
            const float myOriQy = granData->oriQy[myOwnerID];
            const float myOriQz = granData->oriQz[myOwnerID];
            applyOriQToVector3<float, deme::oriQ_t>(myRelPosX, myRelPosY, myRelPosZ, myOriQw, myOriQx, myOriQy,
                                                    myOriQz);
            // The bin number that I live in (with fractions)?
            myPosX = ownerX + (double)myRelPosX;
            myPosY = ownerY + (double)myRelPosY;
            myPosZ = ownerZ + (double)myRelPosZ;
            double myBinX = myPosX / simParams->binSize;
            double myBinY = myPosY / simParams->binSize;
            double myBinZ = myPosZ / simParams->binSize;
            // How many bins my radius spans (with fractions)?
            double myRadiusSpan = myRadius / simParams->binSize;
            // printf("myRadius: %f\n", myRadiusSpan);
            // Now, figure out how many bins I touch in each direction
            deme::binsSphereTouches_t numX =
                (unsigned int)(myBinX + myRadiusSpan) - (unsigned int)(myBinX - myRadiusSpan) + 1;
            deme::binsSphereTouches_t numY =
                (unsigned int)(myBinY + myRadiusSpan) - (unsigned int)(myBinY - myRadiusSpan) + 1;
            deme::binsSphereTouches_t numZ =
                (unsigned int)(myBinZ + myRadiusSpan) - (unsigned int)(myBinZ - myRadiusSpan) + 1;
            // TODO: Add an error message if numX * numY * numZ > MAX(binsSphereTouches_t)

            // Write the number of bins this sphere touches back to the global array
            numBinsSphereTouches[triID] = numX * numY * numZ;
            // printf("This sp takes num of bins: %u\n", numX * numY * numZ);
        }

        // Each sphere entity should also check if it overlaps with an analytical boundary-type geometry
        for (deme::objID_t objB = 0; objB < simParams->nAnalGM; objB++) {
            deme::contact_t contact_type;
            deme::bodyID_t objBOwner = objOwner[objB];
            // Grab family number from memory (not jitified: b/c family number can change frequently in a sim)
            unsigned int objFamilyNum = granData->familyID[objBOwner];
            unsigned int maskMatID =
                locateMaskPair<unsigned int>((unsigned int)triFamilyNum, (unsigned int)objFamilyNum);
            // If marked no contact, skip ths iteration
            if (granData->familyMasks[maskMatID] != deme::DONT_PREVENT_CONTACT) {
                continue;
            }
            double ownerX, ownerY, ownerZ;
            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[objBOwner], granData->locX[objBOwner],
                granData->locY[objBOwner], granData->locZ[objBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            const float ownerOriQw = granData->oriQw[objBOwner];
            const float ownerOriQx = granData->oriQx[objBOwner];
            const float ownerOriQy = granData->oriQy[objBOwner];
            const float ownerOriQz = granData->oriQz[objBOwner];
            float objBRelPosX = objRelPosX[objB];
            float objBRelPosY = objRelPosY[objB];
            float objBRelPosZ = objRelPosZ[objB];
            float objBRotX = objRotX[objB];
            float objBRotY = objRotY[objB];
            float objBRotZ = objRotZ[objB];
            applyOriQToVector3<float, deme::oriQ_t>(objBRelPosX, objBRelPosY, objBRelPosZ, ownerOriQw, ownerOriQx,
                                                    ownerOriQy, ownerOriQz);
            applyOriQToVector3<float, deme::oriQ_t>(objBRotX, objBRotY, objBRotZ, ownerOriQw, ownerOriQx, ownerOriQy,
                                                    ownerOriQz);
            double objBPosX = ownerX + (double)objBRelPosX;
            double objBPosY = ownerY + (double)objBRelPosY;
            double objBPosZ = ownerZ + (double)objBRelPosZ;
            contact_type = checkSphereEntityOverlap<double>(
                myPosX, myPosY, myPosZ, myRadius, objType[objB], objBPosX, objBPosY, objBPosZ, objBRotX, objBRotY,
                objBRotZ, objSize1[objB], objSize2[objB], objSize3[objB], objNormal[objB], simParams->beta);

            if (contact_type) {
                contact_count++;
            }
        }
        numAnalGeoSphereTouches[triID] = contact_count;
    }
}

__global__ void populateBinTriangleTouchingPairs(deme::DEMSimParams* simParams,
                                                 deme::DEMDataKT* granData,
                                                 deme::binsTriangleTouchPairs_t* numBinTriTouchesScan,
                                                 deme::binID_t* binIDsEachTriTouches,
                                                 deme::triID_t* triIDsEachBinTouches) {
    deme::bodyID_t triID = blockIdx.x * blockDim.x + threadIdx.x;
    if (triID < simParams->nSpheresGM) {
        double myPosX, myPosY, myPosZ;
        double myRadius;
        unsigned int triFamilyNum;
        deme::binSphereTouchPairs_t mySphereGeoReportOffset = numAnalGeoSphereTouchesScan[triID];
        {
            // My sphere voxel ID and my relPos
            deme::bodyID_t myOwnerID = granData->ownerClumpBody[triID];
            triFamilyNum = granData->familyID[myOwnerID];
            float myRelPosX, myRelPosY, myRelPosZ;
            double ownerX, ownerY, ownerZ;

            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPosXYZ, myRadius (in CD kernels, radius needs to be expanded)
            // Use an input named exactly `triID' which is the id of this sphere component
            {
                _componentAcqStrat_;
                myRadius += simParams->beta;
            }

            // Get the offset of my spot where I should start writing back to the global bin--sphere pair registration
            // array
            deme::binSphereTouchPairs_t myReportOffset = numBinsSphereTouchesScan[triID];
            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[myOwnerID], granData->locX[myOwnerID],
                granData->locY[myOwnerID], granData->locZ[myOwnerID], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            const float myOriQw = granData->oriQw[myOwnerID];
            const float myOriQx = granData->oriQx[myOwnerID];
            const float myOriQy = granData->oriQy[myOwnerID];
            const float myOriQz = granData->oriQz[myOwnerID];
            applyOriQToVector3<float, deme::oriQ_t>(myRelPosX, myRelPosY, myRelPosZ, myOriQw, myOriQx, myOriQy,
                                                    myOriQz);
            // The bin number that I live in (with fractions)?
            myPosX = ownerX + (double)myRelPosX;
            myPosY = ownerY + (double)myRelPosY;
            myPosZ = ownerZ + (double)myRelPosZ;
            double myBinX = myPosX / simParams->binSize;
            double myBinY = myPosY / simParams->binSize;
            double myBinZ = myPosZ / simParams->binSize;
            // How many bins my radius spans (with fractions)?
            double myRadiusSpan = myRadius / simParams->binSize;
            // Now, write the IDs of those bins that I touch, back to the global memory
            deme::binID_t thisBinID;
            for (unsigned int k = (unsigned int)(myBinZ - myRadiusSpan); k <= (unsigned int)(myBinZ + myRadiusSpan);
                 k++) {
                for (unsigned int j = (unsigned int)(myBinY - myRadiusSpan); j <= (unsigned int)(myBinY + myRadiusSpan);
                     j++) {
                    for (unsigned int i = (unsigned int)(myBinX - myRadiusSpan);
                         i <= (unsigned int)(myBinX + myRadiusSpan); i++) {
                        thisBinID = (deme::binID_t)i + (deme::binID_t)j * simParams->nbX +
                                    (deme::binID_t)k * simParams->nbX * simParams->nbY;
                        binIDsEachSphereTouches[myReportOffset] = thisBinID;
                        sphereIDsEachBinTouches[myReportOffset] = triID;
                        myReportOffset++;
                    }
                }
            }
        }

        // Each sphere entity should also check if it overlaps with an analytical boundary-type geometry
        for (deme::objID_t objB = 0; objB < simParams->nAnalGM; objB++) {
            deme::contact_t contact_type;
            deme::bodyID_t objBOwner = objOwner[objB];
            // Grab family number from memory (not jitified: b/c family number can change frequently in a sim)
            unsigned int objFamilyNum = granData->familyID[objBOwner];
            unsigned int maskMatID = locateMaskPair<unsigned int>(triFamilyNum, objFamilyNum);
            // If marked no contact, skip ths iteration
            if (granData->familyMasks[maskMatID] != deme::DONT_PREVENT_CONTACT) {
                continue;
            }
            double ownerX, ownerY, ownerZ;
            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[objBOwner], granData->locX[objBOwner],
                granData->locY[objBOwner], granData->locZ[objBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            const float ownerOriQw = granData->oriQw[objBOwner];
            const float ownerOriQx = granData->oriQx[objBOwner];
            const float ownerOriQy = granData->oriQy[objBOwner];
            const float ownerOriQz = granData->oriQz[objBOwner];
            float objBRelPosX = objRelPosX[objB];
            float objBRelPosY = objRelPosY[objB];
            float objBRelPosZ = objRelPosZ[objB];
            float objBRotX = objRotX[objB];
            float objBRotY = objRotY[objB];
            float objBRotZ = objRotZ[objB];
            applyOriQToVector3<float, deme::oriQ_t>(objBRelPosX, objBRelPosY, objBRelPosZ, ownerOriQw, ownerOriQx,
                                                    ownerOriQy, ownerOriQz);
            applyOriQToVector3<float, deme::oriQ_t>(objBRotX, objBRotY, objBRotZ, ownerOriQw, ownerOriQx, ownerOriQy,
                                                    ownerOriQz);
            double objBPosX = ownerX + (double)objBRelPosX;
            double objBPosY = ownerY + (double)objBRelPosY;
            double objBPosZ = ownerZ + (double)objBRelPosZ;
            contact_type = checkSphereEntityOverlap<double>(
                myPosX, myPosY, myPosZ, myRadius, objType[objB], objBPosX, objBPosY, objBPosZ, objBRotX, objBRotY,
                objBRotZ, objSize1[objB], objSize2[objB], objSize3[objB], objNormal[objB], simParams->beta);

            if (contact_type) {
                idGeoA[mySphereGeoReportOffset] = triID;
                idGeoB[mySphereGeoReportOffset] = (deme::bodyID_t)objB;
                contactType[mySphereGeoReportOffset] = contact_type;
                mySphereGeoReportOffset++;
            }
        }
    }
}
