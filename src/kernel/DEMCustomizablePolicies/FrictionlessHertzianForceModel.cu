#include "hip/hip_runtime.h"
// DEM force calculation strategies, modifiable

// Material properties and time (user referrable)
float E_cnt, CoR_cnt;
{
    mat2ProxyContactParam<float>(E_cnt, CoR_cnt, E[bodyAMatType], nu[bodyAMatType], CoR[bodyAMatType], E[bodyBMatType],
                                 nu[bodyBMatType], CoR[bodyBMatType]);
}

float3 rotVelCPA, rotVelCPB;
{
    // We also need the relative velocity between A and B in global frame to use in the damping terms
    // To get that, we need contact points' rotational velocity in GLOBAL frame
    // This is local rotational velocity (the portion of linear vel contributed by rotation)
    rotVelCPA = cross(ARotVel, locCPA);
    rotVelCPB = cross(BRotVel, locCPB);
    // This is mapping from local rotational velocity to global
    applyOriQToVector3<float, smug::oriQ_t>(rotVelCPA.x, rotVelCPA.y, rotVelCPA.z, AoriQw, AoriQx, AoriQy, AoriQz);
    applyOriQToVector3<float, smug::oriQ_t>(rotVelCPB.x, rotVelCPB.y, rotVelCPB.z, BoriQw, BoriQx, BoriQy, BoriQz);
}

// The (total) relative linear velocity of A relative to B
const float3 velB2A = (ALinVel + rotVelCPA) - (BLinVel + rotVelCPB);
const float projection = dot(velB2A, B2A);

const float mass_eff = (AOwnerMass * BOwnerMass) / (AOwnerMass + BOwnerMass);
float sqrt_Rd = sqrt(overlapDepth * (ARadius * BRadius) / (ARadius + BRadius));
const float Sn = 2. * E_cnt * sqrt_Rd;

const float loge = (CoR_cnt < SMUG_DEM_TINY_FLOAT) ? log(SMUG_DEM_TINY_FLOAT) : log(CoR_cnt);
float beta = loge / sqrt(loge * loge + smug::PI_SQUARED);

const float k_n = smug::TWO_OVER_THREE * Sn;
const float gamma_n = smug::TWO_TIMES_SQRT_FIVE_OVER_SIX * beta * sqrt(Sn * mass_eff);

// normal force (that A feels)
force += (k_n * overlapDepth + gamma_n * projection) * B2A;