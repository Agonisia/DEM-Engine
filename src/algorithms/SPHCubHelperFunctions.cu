#include "hip/hip_runtime.h"
#include <algorithms/SPHCubHelperFunctions.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/util_debug.cuh>

// CustomMin functor
struct CustomCubFloat3Add {
    HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__ __host__ float3 operator()(const float3& a, const float3& b) const {
        return ::make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
    }
};

void PrefixScanExclusiveCub(std::vector<int, sgps::ManagedAllocator<int>>& d_in,
                            std::vector<int, sgps::ManagedAllocator<int>>& d_out) {
    d_out.resize(d_in.size());
    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in.data(), d_out.data(), d_in.size());
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in.data(), d_out.data(), d_in.size());

    hipDeviceSynchronize();
}

void PairRadixSortAscendCub(std::vector<int, sgps::ManagedAllocator<int>>& d_keys_in,
                            std::vector<int, sgps::ManagedAllocator<int>>& d_keys_out,
                            std::vector<int, sgps::ManagedAllocator<int>>& d_values_in,
                            std::vector<int, sgps::ManagedAllocator<int>>& d_values_out) {
    d_keys_out.resize(d_keys_in.size());
    d_values_out.resize(d_values_in.size());

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_keys_out.data(),
                                    d_values_in.data(), d_values_out.data(), d_keys_in.size());
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_keys_out.data(),
                                    d_values_in.data(), d_values_out.data(), d_keys_in.size());

    hipDeviceSynchronize();
}

void PairRadixSortAscendCub(std::vector<int, sgps::ManagedAllocator<int>>& d_keys_in,
                            std::vector<int, sgps::ManagedAllocator<int>>& d_keys_out,
                            std::vector<float, sgps::ManagedAllocator<float>>& d_values_in,
                            std::vector<float, sgps::ManagedAllocator<float>>& d_values_out) {
    d_keys_out.resize(d_keys_in.size());
    d_values_out.resize(d_values_in.size());

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_keys_out.data(),
                                    d_values_in.data(), d_values_out.data(), d_keys_in.size());
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_keys_out.data(),
                                    d_values_in.data(), d_values_out.data(), d_keys_in.size());

    hipDeviceSynchronize();
}

void PairRadixSortAscendCub(std::vector<int, sgps::ManagedAllocator<int>>& d_keys_in,
                            std::vector<int, sgps::ManagedAllocator<int>>& d_keys_out,
                            std::vector<float3, sgps::ManagedAllocator<float3>>& d_values_in,
                            std::vector<float3, sgps::ManagedAllocator<float3>>& d_values_out) {
    d_keys_out.resize(d_keys_in.size());
    d_values_out.resize(d_values_in.size());

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_keys_out.data(),
                                    d_values_in.data(), d_values_out.data(), d_keys_in.size());
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_keys_out.data(),
                                    d_values_in.data(), d_values_out.data(), d_keys_in.size());

    hipDeviceSynchronize();
}

void RunLengthEncodeCub(std::vector<int, sgps::ManagedAllocator<int>>& d_in,
                        std::vector<int, sgps::ManagedAllocator<int>>& d_unique_out,
                        std::vector<int, sgps::ManagedAllocator<int>>& d_counts_out) {
    std::vector<int, sgps::ManagedAllocator<int>> d_num_runs_out;
    d_unique_out.resize(d_in.size());
    d_counts_out.resize(d_in.size());
    d_num_runs_out.resize(1);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage, temp_storage_bytes, d_in.data(), d_unique_out.data(),
                                       d_counts_out.data(), d_num_runs_out.data(), d_in.size());
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run encoding
    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage, temp_storage_bytes, d_in.data(), d_unique_out.data(),
                                       d_counts_out.data(), d_num_runs_out.data(), d_in.size());

    hipDeviceSynchronize();

    d_unique_out.erase(d_unique_out.begin() + d_num_runs_out[0], d_unique_out.end());
    d_counts_out.erase(d_counts_out.begin() + d_num_runs_out[0], d_counts_out.end());

    hipDeviceSynchronize();
}

void SumReduceByKeyCub(std::vector<int, sgps::ManagedAllocator<int>>& d_keys_in,
                       std::vector<int, sgps::ManagedAllocator<int>>& d_unique_out,
                       std::vector<float, sgps::ManagedAllocator<float>>& d_values_in,
                       std::vector<float, sgps::ManagedAllocator<float>>& d_aggregates_out) {
    d_unique_out.resize(d_keys_in.size());
    d_aggregates_out.resize(d_values_in.size());

    std::vector<int, sgps::ManagedAllocator<int>> d_num_runs_out;
    d_num_runs_out.resize(1);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_unique_out.data(),
                                   d_values_in.data(), d_aggregates_out.data(), d_num_runs_out.data(), hipcub::Sum(),
                                   d_keys_in.size());
    hipDeviceSynchronize();

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_unique_out.data(),
                                   d_values_in.data(), d_aggregates_out.data(), d_num_runs_out.data(), hipcub::Sum(),
                                   d_keys_in.size());
    hipDeviceSynchronize();

    d_unique_out.erase(d_unique_out.begin() + d_num_runs_out[0], d_unique_out.end());
    d_aggregates_out.erase(d_aggregates_out.begin() + d_num_runs_out[0], d_aggregates_out.end());
    hipDeviceSynchronize();
}

void SumReduceByKeyCub(std::vector<int, sgps::ManagedAllocator<int>>& d_keys_in,
                       std::vector<int, sgps::ManagedAllocator<int>>& d_unique_out,
                       std::vector<float3, sgps::ManagedAllocator<float3>>& d_values_in,
                       std::vector<float3, sgps::ManagedAllocator<float3>>& d_aggregates_out) {
    d_unique_out.resize(d_keys_in.size());
    d_aggregates_out.resize(d_values_in.size());

    std::vector<int, sgps::ManagedAllocator<int>> d_num_runs_out;
    d_num_runs_out.resize(1);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_unique_out.data(),
                                   d_values_in.data(), d_aggregates_out.data(), d_num_runs_out.data(),
                                   CustomCubFloat3Add(), d_keys_in.size());
    hipDeviceSynchronize();

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in.data(), d_unique_out.data(),
                                   d_values_in.data(), d_aggregates_out.data(), d_num_runs_out.data(),
                                   CustomCubFloat3Add(), d_keys_in.size());
    hipDeviceSynchronize();

    d_unique_out.erase(d_unique_out.begin() + d_num_runs_out[0], d_unique_out.end());
    d_aggregates_out.erase(d_aggregates_out.begin() + d_num_runs_out[0], d_aggregates_out.end());
    hipDeviceSynchronize();
}