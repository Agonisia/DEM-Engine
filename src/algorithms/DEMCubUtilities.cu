//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//  All rights reserved.

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>
#include <algorithms/DEMCubBasedSubroutines.h>

#include <core/utils/GpuError.h>
#include <algorithms/DEMCubWrappers.cu>

namespace sgps {

void sumReduce(float* d_in, float* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMSum<float, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void flagMaxReduce(notStupidBool_t* d_in,
                   notStupidBool_t* d_out,
                   size_t n,
                   hipStream_t& this_stream,
                   DEMSolverStateData& scratchPad) {
    cubDEMMax<notStupidBool_t, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

}  // namespace sgps
