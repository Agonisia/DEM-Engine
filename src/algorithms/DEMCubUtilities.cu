//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//
//	SPDX-License-Identifier: BSD-3-Clause

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>
#include <algorithms/DEMCubBasedSubroutines.h>

#include <core/utils/GpuError.h>
#include <algorithms/DEMCubWrappers.cu>

namespace sgps {

// These functions interconnecting the cub-part and cpp-part of the code cannot be templated... because of cmake
// restrictions. Not much that I can do, other than writing them all out.

void doubleSumReduce(double* d_in, double* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMSum<double, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void floatSumReduce(float* d_in, float* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMSum<float, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void boolMaxReduce(notStupidBool_t* d_in,
                   notStupidBool_t* d_out,
                   size_t n,
                   hipStream_t& this_stream,
                   DEMSolverStateData& scratchPad) {
    cubDEMMax<notStupidBool_t, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void floatMaxReduce(float* d_in, float* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMMax<float, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void floatMaxReduceByKey(notStupidBool_t* d_keys_in,
                         notStupidBool_t* d_unique_out,
                         float* d_vals_in,
                         float* d_aggregates_out,
                         size_t* d_num_out,
                         size_t n,
                         hipStream_t& this_stream,
                         DEMSolverStateData& scratchPad) {
    // I'm not sure how to pass cuda hipcub::Sum() as a template argument here, so I used a custom add...
    CubFloatAdd add_op;
    cubDEMReduceByKeys<notStupidBool_t, float, CubFloatAdd, DEMSolverStateData>(
        d_keys_in, d_unique_out, d_vals_in, d_aggregates_out, d_num_out, add_op, n, this_stream, scratchPad);
}

}  // namespace sgps
