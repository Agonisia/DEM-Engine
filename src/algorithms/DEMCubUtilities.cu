//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//  All rights reserved.

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>
#include <algorithms/DEMCubBasedSubroutines.h>

#include <core/utils/GpuError.h>
#include <algorithms/DEMCubWrappers.cu>

namespace sgps {

void sumReduce(float* d_in, float* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateDataDT& scratchPad) {
    cubDEMSum<float, DEMSolverStateDataDT>(d_in, d_out, n, this_stream, scratchPad);
}

}  // namespace sgps
