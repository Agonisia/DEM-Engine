//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//  All rights reserved.

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>

#include <algorithms/DEMCubHelperFunctions.h>

#include <core/utils/GpuError.h>

namespace sgps {

void cubPrefixScan(sgps::binsSphereTouches_t* d_in,
                   sgps::binsSphereTouchesScan_t* d_out,
                   size_t n,
                   GpuManager::StreamInfo& streamInfo,
                   sgps::DEMSolverStateData& scratchPad) {
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(NULL, temp_storage_bytes, d_in, d_out, n);
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_scratch_space, temp_storage_bytes, d_in, d_out, n);
    GPU_CALL(hipStreamSynchronize(streamInfo.stream));
}

void cubSortByKeys(sgps::binID_t* d_keys,
                   sgps::bodyID_t* d_vals,
                   size_t n,
                   GpuManager::StreamInfo& streamInfo,
                   sgps::DEMSolverStateData& scratchPad) {
    /*
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(NULL, temp_storage_bytes, d_keys, d_vals, n, 0,
    sizeof(sgps::binID_t)*BITS_PER_BYTE,streamInfo.stream,false); void* d_scratch_space =
    (void*)scratchPad.allocateScratchSpace(temp_storage_bytes); hipcub::DeviceRadixSort::SortPairs(d_scratch_space,
    temp_storage_bytes, d_keys, d_vals, n, 0, sizeof(sgps::binID_t)*BITS_PER_BYTE,streamInfo.stream,false);
    GPU_CALL(hipStreamSynchronize(streamInfo.stream));
    */
}

}  // namespace sgps
