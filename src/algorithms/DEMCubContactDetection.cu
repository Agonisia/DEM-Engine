//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//  All rights reserved.

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>

#include <algorithms/DEMCubHelperFunctions.h>

#include <core/utils/GpuError.h>

namespace sgps {

// template <typename T1, typename T2>
// void cubPrefixScan(T1* d_in,
//                    T2* d_out,
//                    size_t n,
//                    hipStream_t& this_stream,
//                    DEMSolverStateData& scratchPad) {

void cubPrefixScan_binSphere(binsSphereTouches_t* d_in,
                             binSphereTouchPairs_t* d_out,
                             size_t n,
                             hipStream_t& this_stream,
                             DEMSolverStateDataKT& scratchPad) {
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(NULL, cub_scratch_bytes, d_in, d_out, n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_scratch_space, cub_scratch_bytes, d_in, d_out, n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
}

void cubPrefixScan_contacts(spheresBinTouches_t* d_in,
                            contactPairs_t* d_out,
                            size_t n,
                            hipStream_t& this_stream,
                            DEMSolverStateDataKT& scratchPad) {
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(NULL, cub_scratch_bytes, d_in, d_out, n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_scratch_space, cub_scratch_bytes, d_in, d_out, n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
}

void cubSortByKeys(binID_t* d_keys_in,
                   binID_t* d_keys_out,
                   bodyID_t* d_vals_in,
                   bodyID_t* d_vals_out,
                   size_t n,
                   hipStream_t& this_stream,
                   DEMSolverStateDataKT& scratchPad) {
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(NULL, cub_scratch_bytes, d_keys_in, d_keys_out, d_vals_in, d_vals_out, n, 0,
                                    sizeof(sgps::binID_t) * BITS_PER_BYTE, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes);
    hipcub::DeviceRadixSort::SortPairs(d_scratch_space, cub_scratch_bytes, d_keys_in, d_keys_out, d_vals_in, d_vals_out, n,
                                    0, sizeof(sgps::binID_t) * BITS_PER_BYTE, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
}

void cubUnique(binID_t* d_in,
               binID_t* d_out,
               size_t* d_num_out,
               size_t n,
               hipStream_t& this_stream,
               DEMSolverStateDataKT& scratchPad) {
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceSelect::Unique(NULL, cub_scratch_bytes, d_in, d_out, d_num_out, n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes);
    hipcub::DeviceSelect::Unique(d_scratch_space, cub_scratch_bytes, d_in, d_out, d_num_out, n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
}

void cubRunLengthEncode(binID_t* d_in,
                        binID_t* d_unique_out,
                        spheresBinTouches_t* d_counts_out,
                        size_t* d_num_out,
                        size_t n,
                        hipStream_t& this_stream,
                        DEMSolverStateDataKT& scratchPad) {
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceRunLengthEncode::Encode(NULL, cub_scratch_bytes, d_in, d_unique_out, d_counts_out, d_num_out, n,
                                       this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes);
    hipcub::DeviceRunLengthEncode::Encode(d_scratch_space, cub_scratch_bytes, d_in, d_unique_out, d_counts_out, d_num_out,
                                       n, this_stream, false);
    GPU_CALL(hipStreamSynchronize(this_stream));
}

}  // namespace sgps
