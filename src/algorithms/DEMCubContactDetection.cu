//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//  All rights reserved.

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>

#include <algorithms/DEMCubHelperFunctions.h>

#include <core/utils/GpuError.h>

namespace sgps {

void cubPrefixScan(binsSphereTouches_t* d_in,
                   binsSphereTouchesScan_t* d_out,
                   size_t n,
                   GpuManager::StreamInfo& streamInfo,
                   DEMSolverStateData& scratchPad) {
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(NULL, cub_scratch_bytes, d_in, d_out, n, streamInfo.stream, false);
    GPU_CALL(hipStreamSynchronize(streamInfo.stream));
    void* d_scratch_space = (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_scratch_space, cub_scratch_bytes, d_in, d_out, n, streamInfo.stream, false);
    GPU_CALL(hipStreamSynchronize(streamInfo.stream));
}

void cubSortByKeys(binID_t* d_keys,
                   bodyID_t* d_vals,
                   size_t n,
                   GpuManager::StreamInfo& streamInfo,
                   DEMSolverStateData& scratchPad) {
    /*
    size_t cub_scratch_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(NULL, cub_scratch_bytes, d_keys, d_vals, n, 0,
    sizeof(sgps::binID_t)*BITS_PER_BYTE,streamInfo.stream,false); void* d_scratch_space =
    (void*)scratchPad.allocateScratchSpace(cub_scratch_bytes); hipcub::DeviceRadixSort::SortPairs(d_scratch_space,
    cub_scratch_bytes, d_keys, d_vals, n, 0, sizeof(sgps::binID_t)*BITS_PER_BYTE,streamInfo.stream,false);
    GPU_CALL(hipStreamSynchronize(streamInfo.stream));
    */
}

}  // namespace sgps
