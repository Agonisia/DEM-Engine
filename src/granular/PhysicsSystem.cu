//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//  All rights reserved.

#include <chrono>
#include <cstring>
#include <hip/hip_runtime.h>
#include <granular/PhysicsSystem.h>
#include <iostream>
#include <thread>

namespace sgps {

void kinematicThread::operator()() {
    // Set the device for this thread
    hipSetDevice(streamInfo.device);

    // run a while loop producing stuff in each iteration;
    // once produced, it should be made available to the dynamic via memcpy
    while (!pSchedSupport->dynamicDone) {
        // before producing something, a new work order should be in place. Wait on
        // it
        if (!pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh) {
            pSchedSupport->schedulingStats.nTimesKinematicHeldBack++;
            std::unique_lock<std::mutex> lock(pSchedSupport->kinematicCanProceed);
            while (!pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh) {
                // loop to avoid spurious wakeups
                pSchedSupport->cv_KinematicCanProceed.wait(lock);
            }
            // getting here means that new "work order" data has been provided
            {
                // acquire lock and supply the dynamic with fresh produce
                std::lock_guard<std::mutex> lock(pSchedSupport->kinematicOwnedBuffer_AccessCoordination);
                hipMemcpy(voxelID.data(), transferBuffer_voxelID.data(), N_INPUT_ITEMS * sizeof(voxelID_default_t),
                           hipMemcpyDeviceToDevice);
            }
        }

        // figure out the amount of shared mem
        // hipDeviceGetAttribute.hipDeviceAttributeMaxSharedMemoryPerBlock

        // produce something here; fake stuff for now
        // hipStream_t currentStream;
        // hipStreamCreate(&currentStream);

        auto data_arg = voxelID.data();
        void* args[] = {(void*)(&data_arg)};
        // hipLaunchKernel((void*)&kinematicTestKernel, dim3(1), dim3(N_INPUT_ITEMS), args, 0, streamInfo.stream);
        // kinematicTestKernel<<<1, 4, 0, kinematicStream.stream>>>(voxelID.data());
        hipDeviceSynchronize();
        // hipStreamDestroy(currentStream);

        /* for reference staff
        for (int j = 0; j < N_MANUFACTURED_ITEMS; j++) {
            // kinematicTestKernel<<<1, 1, 0, kinematicStream.stream>>>();

            // use hipLaunchKernel
            // hipLaunchKernel((void*)&kinematicTestKernel, dim3(1), dim3(1), NULL, 0, stream_id);
            // example argument list:
            //  args = { &arg1, &arg2, ... &argN };
            // hipLaunchKernel((void*)&kinematicTestKernelWithArgs, dim3(1), dim3(1), &args, 0, stream_id);
            kinematicTestKernel<<<1, 1>>>();
            hipDeviceSynchronize();

            int indx = j % N_INPUT_ITEMS;
            product[j] += this->costlyProductionStep(j) + inputData[indx];
        }
        */

        // make it clear that the data for most recent work order has
        // been used, in case there is interest in updating it
        pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh = false;

        {
            // acquire lock and supply the dynamic with fresh produce
            std::lock_guard<std::mutex> lock(pSchedSupport->dynamicOwnedBuffer_AccessCoordination);
            hipMemcpy(pDynamicOwnedBuffer_voxelID, voxelID.data(), N_MANUFACTURED_ITEMS * sizeof(voxelID_default_t),
                       hipMemcpyDeviceToDevice);
        }
        pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh = true;
        pSchedSupport->schedulingStats.nDynamicUpdates++;

        // signal the dynamic that it has fresh produce
        pSchedSupport->cv_DynamicCanProceed.notify_all();
    }

    // in case the dynamic is hanging in there...
    pSchedSupport->cv_DynamicCanProceed.notify_all();
}

void dynamicThread::operator()() {
    // Set the gpu for this thread
    hipSetDevice(streamInfo.device);

    // acquire lock to prevent the kinematic to mess up
    // with the transfer buffer while the latter is used

    for (int cycle = 0; cycle < nDynamicCycles; cycle++) {
        // if the produce is fresh, use it
        if (pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh) {
            {
                // acquire lock and use the content of the dynamic-owned transfer
                // buffer
                std::lock_guard<std::mutex> lock(pSchedSupport->dynamicOwnedBuffer_AccessCoordination);
                hipMemcpy(voxelID.data(), transferBuffer_voxelID.data(),
                           N_MANUFACTURED_ITEMS * sizeof(voxelID_default_t), hipMemcpyDeviceToDevice);
            }
            pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh = false;
            pSchedSupport->stampLastUpdateOfDynamic = cycle;
        }

        // if it's the case, it's important at this point to let the kinematic know
        // that this is the last dynamic cycle; this is important otherwise the
        // kinematic will hang waiting for communication swith the dynamic
        if (cycle == (nDynamicCycles - 1))
            pSchedSupport->dynamicDone = true;

        // if the kinematic is idle, give it the opportunity to get busy again
        if (!pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh) {
            // acquire lock and refresh the work order for the kinematic
            {
                std::lock_guard<std::mutex> lock(pSchedSupport->kinematicOwnedBuffer_AccessCoordination);
                hipMemcpy(pKinematicOwnedBuffer_voxelID, voxelID.data(), N_INPUT_ITEMS * sizeof(voxelID_default_t),
                           hipMemcpyDeviceToDevice);
            }
            pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh = true;
            pSchedSupport->schedulingStats.nKinematicUpdates++;
            // signal the kinematic that it has data for a new work order
            pSchedSupport->cv_KinematicCanProceed.notify_all();
        }

        /* Currently no work at all
        // this is the fake place where produce is used;
        for (int j = 0; j < N_MANUFACTURED_ITEMS; j++) {
            outcome[j] += this->localUse(j);
        }
        */

        int totGPU;
        hipGetDeviceCount(&totGPU);
        printf("Total device: %d\n", totGPU);

        std::cout << "Dynamic side values. Cycle: " << cycle << std::endl;

        // dynamic wrapped up one cycle
        pSchedSupport->currentStampOfDynamic++;

        // check if we need to wait; i.e., if dynamic drifted too much into future, then we must wait a bit before the
        // next cycle begins
        if (pSchedSupport->dynamicShouldWait()) {
            // wait for a signal from the kinematic to indicate that
            // the kinematic has caught up
            pSchedSupport->schedulingStats.nTimesDynamicHeldBack++;
            std::unique_lock<std::mutex> lock(pSchedSupport->dynamicCanProceed);
            while (!pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh) {
                // loop to avoid spurious wakeups
                pSchedSupport->cv_DynamicCanProceed.wait(lock);
            }
        }
    }
}

int dynamicThread::localUse(int val) {
    hipSetDevice(streamInfo.device);
    // std::this_thread::sleep_for(std::chrono::milliseconds(dynamicAverageTime));

    // dynamicTestKernel<<<1, 1>>>();
    // hipLaunchKernel((void*)&dynamicTestKernel, dim3(1), dim3(1), NULL, 0, streamInfo.stream);
    // hipDeviceSynchronize();
    return 2 * val;
}

void kinematicThread::primeDynamic() {
    // transfer produce to dynamic buffer
    hipMemcpy(pDynamicOwnedBuffer_voxelID, voxelID.data(), N_INPUT_ITEMS * sizeof(voxelID_default_t),
               hipMemcpyDeviceToDevice);
    pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh = true;
    pSchedSupport->schedulingStats.nDynamicUpdates++;
}

}  // namespace sgps
