#include "hip/hip_runtime.h"
#include <chrono>
#include <cstring>
#include <hip/hip_runtime.h>
#include <granular/PhysicsSystem.h>
#include <iostream>
#include <thread>

namespace sgps {

__global__ void dynamicTestKernel() { printf("Dynamic run\n"); }
__global__ void kinematicTestKernel() { printf("Kinematic run\n"); }

void kinematicThread::operator()() {
  // run a while loop producing stuff in each iteration;
  // once produced, it should be made available to the dynamic via memcpy
  while (!pSchedSupport->dynamicDone) {
    // before producing something, a new work order should be in place. Wait on
    // it
    if (!pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh) {
      pSchedSupport->schedulingStats.nTimesKinematicHeldBack++;
      std::unique_lock<std::mutex> lock(pSchedSupport->kinematicCanProceed);
      while (!pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh) {
        // loop to avoid spurious wakeups
        pSchedSupport->cv_KinematicCanProceed.wait(lock);
      }
      // getting here means that new "work order" data has been provided
      {
        // acquire lock and supply the dynamic with fresh produce
        std::lock_guard<std::mutex> lock(
            pSchedSupport->kinematicOwnedBuffer_AccessCoordination);
        memcpy(inputData, transferBuffer, N_INPUT_ITEMS * sizeof(int));
      }
    }

    int totGPU;
    hipGetDeviceCount(&totGPU);
    printf("Total device: %d\n", totGPU);

    // produce something here; fake stuff for now
    for (int j = 0; j < N_MANUFACTURED_ITEMS; j++) {
      kinematicTestKernel<<<1, 1>>>();
      hipDeviceSynchronize();

      int indx = j % N_INPUT_ITEMS;
      product[j] += this->costlyProductionStep(j) + inputData[indx];
    }

    // make it clear that the data for most recent work order has
    // been used, in case there is interest in updating it
    pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh = false;

    {
      // acquire lock and supply the dynamic with fresh produce
      std::lock_guard<std::mutex> lock(
          pSchedSupport->dynamicOwnedBuffer_AccessCoordination);
      memcpy(pDynamicOwned_TransfBuffer, product,
             N_MANUFACTURED_ITEMS * sizeof(int));
    }
    pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh = true;
    pSchedSupport->schedulingStats.nDynamicUpdates++;

    // signal the dynamic that it has fresh produce
    pSchedSupport->cv_DynamicCanProceed.notify_all();
  }

  // in case the dynamic is hanging in there...
  pSchedSupport->cv_DynamicCanProceed.notify_all();
}

void dynamicThread::operator()() {
  // acquire lock to prevent the kinematic to mess up
  // with the transfer buffer while the latter is used

  for (int cycle = 0; cycle < nDynamicCycles; cycle++) {
    // if the produce is fresh, use it
    if (pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh) {
      {
        // acquire lock and use the content of the dynamic-owned transfer
        // buffer
        std::lock_guard<std::mutex> lock(
            pSchedSupport->dynamicOwnedBuffer_AccessCoordination);
        memcpy(outcome, transferBuffer, N_MANUFACTURED_ITEMS * sizeof(int));
      }
      pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh = false;
      pSchedSupport->stampLastUpdateOfDynamic = cycle;
    }

    // if it's the case, it's important at this point to let the kinematic know
    // that this is the last dynamic cycle; this is important otherwise the
    // kinematic will hang waiting for communication swith the dynamic
    if (cycle == (nDynamicCycles - 1))
      pSchedSupport->dynamicDone = true;

    // if the kinematic is idle, give it the opportunity to get busy again
    if (!pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh) {
      // acquire lock and refresh the work order for the kinematic
      {
        std::lock_guard<std::mutex> lock(
            pSchedSupport->kinematicOwnedBuffer_AccessCoordination);
        memcpy(pKinematicOwned_TransfBuffer, input4Kinematic,
               N_INPUT_ITEMS * sizeof(int));
      }
      pSchedSupport->kinematicOwned_Cons2ProdBuffer_isFresh = true;
      pSchedSupport->schedulingStats.nKinematicUpdates++;
      // signal the kinematic that it has data for a new work order
      pSchedSupport->cv_KinematicCanProceed.notify_all();
    }

    // this is the fake place where produce is used;
    for (int j = 0; j < N_MANUFACTURED_ITEMS; j++) {
      outcome[j] += this->localUse(j);
    }

    int totGPU;
    hipGetDeviceCount(&totGPU);
    printf("Total device: %d\n", totGPU);

    std::cout << "Dynamic side values. Cycle: " << cycle << std::endl;
    for (int j = 0; j < N_MANUFACTURED_ITEMS; j++) {
      std::cout << outcome[j] << std::endl;
    }

    // dynamic wrapped up one cycle
    pSchedSupport->currentStampOfDynamic++;

    // check if we need to wait; i.e., if dynamic drifted too much into future
    if (pSchedSupport->dynamicShouldWait()) {
      // wait for a signal from the kinematic to indicate that
      // the kinematic has caught up
      pSchedSupport->schedulingStats.nTimesDynamicHeldBack++;
      std::unique_lock<std::mutex> lock(pSchedSupport->dynamicCanProceed);
      while (!pSchedSupport->dynamicOwned_Prod2ConsBuffer_isFresh) {
        // loop to avoid spurious wakeups
        pSchedSupport->cv_DynamicCanProceed.wait(lock);
      }
    }
  }
}

int dynamicThread::localUse(int val) {
  // std::this_thread::sleep_for(std::chrono::milliseconds(dynamicAverageTime));
  dynamicTestKernel<<<1, 1>>>();
  hipDeviceSynchronize();
  return 2 * val;
}

} // namespace sgps
