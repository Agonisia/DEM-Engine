// SPH-DualGPU
// SPH system base class implementation source code

#include <sph/SPHSystem.cuh>
#include <kernel/cudaKernels.cuh>


void SPHSystem::initialize(float radius,vector3 *pos, int n) {
  m_pos = pos;
  m_n = n;

  vector3* vel = new vector3[n];
  vector3* acc = new vector3[n];

  // create a ThreadManager to create a management pool for data sharing between 2 GPUs
  InteractionManager* tm = new InteractionManager();


  // initialize two threads for Dual GPU computation
  kt = new KinematicTread(tm);
  kt->kInitialize(radius, pos, vel, acc, n);
  dt = new DynamicThread(tm);
  dt->dInitialize(radius, pos, vel, acc, n);
}

void SPHSystem::doStepDynamics(float time_step) {
  
  kt->doKinematicStep();
  dt->doDynamicStep();
}

void SPHSystem::printCSV(std::string filename) {
  // create file
  std::ofstream csvFile(filename);

  csvFile << "x_pos,y_pos,z_pos" << std::endl;

  // write particle data into csv file
  for (int i = 0; i < m_n; i++) {
    csvFile << m_pos[i].x << "," << m_pos[i].y << "," << m_pos[i].z
            << std::endl;
  }

  csvFile.close();
}


void KinematicTread::kInitialize(float radius,vector3* pos, vector3* vel, vector3* acc, int n)
{
  k_n = n;
  k_radius = radius;
  hipSetDevice(this->device_id);
  hipMalloc(&k_pos, k_n*sizeof(vector3));
  hipMalloc(&k_vel, k_n*sizeof(vector3));
  hipMalloc(&k_acc, k_n*sizeof(vector3));

  hipMemcpy(k_pos, pos, k_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(k_vel, vel, k_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(k_acc, acc, k_n*sizeof(vector3), hipMemcpyHostToDevice);
}

void KinematicTread::doKinematicStep()
{
  float tolerance = 0.05;
  hipSetDevice(this->device_id);
  //kinematicTestKernel<<<1, 1, 0, kStream>>>();

  // for each step, the kinematic thread needs to do two passes
  // first pass - look for 'number' of potential contacts
  // crate an array to store number of valid potential contacts
  int* cpu_num_arr = new int[k_n];
  for(int i = 0; i<k_n;i++)
  {
    cpu_num_arr[i] = -1;
  }
  int* k_num_arr;
  hipMalloc(&k_num_arr, k_n*sizeof(int));
  hipMemcpy(k_num_arr, cpu_num_arr, k_n*sizeof(int), hipMemcpyHostToDevice);

  // first kinematic pass to calculate offset array
  kinematic1stPass<<<1,k_n,0,kStream>>>(k_pos, k_n, tolerance, k_radius, k_num_arr);

  hipDeviceSynchronize();

  hipMemcpy(cpu_num_arr, k_num_arr, k_n*sizeof(int), hipMemcpyDeviceToHost);

  // calculate the offset array
  int cur_idx = 0;
  int* offset_arr = new int[k_n];
  for(int i = 0; i<k_n;i++)
  {
    offset_arr[i] = cur_idx;
    cur_idx = cur_idx + cpu_num_arr[i];
  }

  int* gpu_offset_arr;
  hipMalloc(&gpu_offset_arr, k_n*sizeof(int));
  hipMemcpy(gpu_offset_arr, offset_arr, k_n*sizeof(int), hipMemcpyHostToDevice);

  // calculate total number of contact
  int contact_sum = 0;

  for(int i = 0; i < k_n; i++)
  {
    contact_sum = contact_sum + cpu_num_arr[i];
  }

  // second kinematic pass to fill the contact pair array
  contactData* cpu_pair_data = new contactData[contact_sum];
  contactData* gpu_pair_data;
  hipMalloc(&gpu_pair_data, contact_sum*sizeof(contactData));
  hipMemcpy(gpu_pair_data, cpu_pair_data, contact_sum*sizeof(contactData), hipMemcpyHostToDevice);

  kinematic2ndPass<<<1,k_n,0,kStream>>>(k_pos,k_n,gpu_offset_arr,k_num_arr,tolerance,k_radius,gpu_pair_data);

  hipMemcpy(cpu_pair_data, gpu_pair_data, contact_sum*sizeof(contactData), hipMemcpyDeviceToHost);

  //std::cout<<"contact pair num: "<<contact_sum<<std::endl;
  //std::cout<<"i: "<<cpu_pair_data[0].contact_pair.x<<", j:"<<cpu_pair_data[0].contact_pair.y<<std::endl;


  // share data through a common ThreadManager instance
  threadManager->contact_pair = cpu_pair_data;
  threadManager->contact_pair_n = contact_sum;

  threadManager->offset = offset_arr;
  threadManager->offset_n = k_n;

  hipDeviceSynchronize();
}

void DynamicThread::dInitialize(float radius,vector3* pos, vector3* vel, vector3* acc, int n)
{
  d_n = n;
  d_radius = radius;
  hipSetDevice(this->device_id);
  hipMalloc(&d_pos, d_n*sizeof(vector3));
  hipMalloc(&d_vel, d_n*sizeof(vector3));
  hipMalloc(&d_acc, d_n*sizeof(vector3));

  hipMemcpy(d_pos, pos, d_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_vel, vel, d_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_acc, acc, d_n*sizeof(vector3), hipMemcpyHostToDevice);
}

void DynamicThread::doDynamicStep()
{
  hipSetDevice(this->device_id);
  //dynamicTestKernel<<<1, 1, 0, dStream>>>();

  // retrieve contact pair data from the ThreadManager
  contactData* cpu_pair = threadManager->contact_pair;
  int cpu_pair_n = threadManager->contact_pair_n;
  
  int* cpu_offset = threadManager->offset;
  int cpu_offset_n = threadManager->offset_n;

  // copy data to the Dynamic GPU
  contactData* gpu_pair_data;
  hipMalloc(&gpu_pair_data, cpu_pair_n*sizeof(contactData));
  hipMemcpy(gpu_pair_data, cpu_pair, cpu_pair_n*sizeof(contactData), hipMemcpyHostToDevice);

  // calculate number of threads needed and number of block needed
  int num_thread = 64;
  int num_block = cpu_pair_n / num_thread + 1;

  dynamicPass<<<num_block, num_thread, 0, dStream>>>(gpu_pair_data, cpu_pair_n, d_pos, d_vel, d_acc, d_radius);

  hipDeviceSynchronize();
}