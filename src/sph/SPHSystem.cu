#include "hip/hip_runtime.h"
// SPH-DualGPU
// SPH system base class implementation source code

#include "SPHSystem.cuh"


void SPHSystem::initialize(float radius,vector3 *pos, int n) {
  m_pos = pos;
  m_n = n;

  vector3* vel = new vector3[n];
  vector3* acc = new vector3[n];

  // create a ThreadManager to create a management pool for data sharing between 2 GPUs
  InteractionManager* tm = new InteractionManager();


  // initialize two threads for Dual GPU computation
  kt = new KinematicTread(tm);
  kt->kInitialize(radius, pos, vel, acc, n);
  dt = new DynamicThread(tm);
  dt->dInitialize(radius, pos, vel, acc, n);
}

void SPHSystem::doStepDynamics(float time_step) {
  
  kt->doKinematicStep();
  dt->doDynamicStep();
}

void SPHSystem::printCSV(std::string filename) {
  // create file
  std::ofstream csvFile(filename);

  csvFile << "x_pos,y_pos,z_pos" << std::endl;

  // write particle data into csv file
  for (int i = 0; i < m_n; i++) {
    csvFile << m_pos[i].x << "," << m_pos[i].y << "," << m_pos[i].z
            << std::endl;
  }

  csvFile.close();
}

// *----------------------------------------
// Kinematic kernals
__global__ void kinematicTestKernel(){
    printf("kinematic run on GPU \n");
  
}

__global__ void kinematic1stPass(vector3* pos, int n, float tolerance,float radius, int* res_arr){
  int idx = threadIdx.x;
  int count = 0;  // count total number of valid contact for the current particle
  if(idx > n){return;}

  for(int i = n-1; i > idx; i--)
  {
    float dist2 = (pos[idx].x - pos[i].x)*(pos[idx].x - pos[i].x)+
    (pos[idx].y - pos[i].y)*(pos[idx].y - pos[i].y)+(pos[idx].z - pos[i].z)*(pos[idx].z - pos[i].z);

    if (dist2 < (radius*2 + tolerance) * (radius*2 + tolerance))
    {
      count++;
    }
  }
  res_arr[idx] = count;
  __syncthreads();
}

__global__ void kinematic2ndPass(vector3* pos, int n, int* offset,int* contact_num_arr,  
                                float tolerance,float radius, contactData* pair_data){
  int idx = threadIdx.x;
  if(contact_num_arr[idx]!=0){
    
    int cur_idx = offset[idx];
    
    for(int i = n-1; i > idx; i--)
    {
      float dist2 = (pos[idx].x - pos[i].x)*(pos[idx].x - pos[i].x)+
      (pos[idx].y - pos[i].y)*(pos[idx].y - pos[i].y)+(pos[idx].z - pos[i].z)*(pos[idx].z - pos[i].z);

      if (dist2 < (radius*2 + tolerance) * (radius*2 + tolerance))
      {
        pair_data[cur_idx].contact_pair.x = idx;
        pair_data[cur_idx].contact_pair.y = i;
        cur_idx++;
      }
    }
  }

  __syncthreads();
  
}

// END of Kinematic kernels
// *----------------------------------------


void KinematicTread::kInitialize(float radius,vector3* pos, vector3* vel, vector3* acc, int n)
{
  k_n = n;
  k_radius = radius;
  hipSetDevice(this->device_id);
  hipMalloc(&k_pos, k_n*sizeof(vector3));
  hipMalloc(&k_vel, k_n*sizeof(vector3));
  hipMalloc(&k_acc, k_n*sizeof(vector3));

  hipMemcpy(k_pos, pos, k_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(k_vel, vel, k_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(k_acc, acc, k_n*sizeof(vector3), hipMemcpyHostToDevice);
}

void KinematicTread::doKinematicStep()
{
  float tolerance = 0.05;
  hipSetDevice(this->device_id);
  //kinematicTestKernel<<<1, 1, 0, kStream>>>();

  // for each step, the kinematic thread needs to do two passes
  // first pass - look for 'number' of potential contacts
  // crate an array to store number of valid potential contacts
  int* cpu_num_arr = new int[k_n];
  for(int i = 0; i<k_n;i++)
  {
    cpu_num_arr[i] = -1;
  }
  int* k_num_arr;
  hipMalloc(&k_num_arr, k_n*sizeof(int));
  hipMemcpy(k_num_arr, cpu_num_arr, k_n*sizeof(int), hipMemcpyHostToDevice);

  // first kinematic pass to calculate offset array
  kinematic1stPass<<<1,k_n,0,kStream>>>(k_pos, k_n, tolerance, k_radius, k_num_arr);

  hipDeviceSynchronize();

  hipMemcpy(cpu_num_arr, k_num_arr, k_n*sizeof(int), hipMemcpyDeviceToHost);

  // calculate the offset array
  int cur_idx = 0;
  int* offset_arr = new int[k_n];
  for(int i = 0; i<k_n;i++)
  {
    offset_arr[i] = cur_idx;
    cur_idx = cur_idx + cpu_num_arr[i];
  }

  int* gpu_offset_arr;
  hipMalloc(&gpu_offset_arr, k_n*sizeof(int));
  hipMemcpy(gpu_offset_arr, offset_arr, k_n*sizeof(int), hipMemcpyHostToDevice);

  // calculate total number of contact
  int contact_sum = 0;

  for(int i = 0; i < k_n; i++)
  {
    contact_sum = contact_sum + cpu_num_arr[i];
  }

  // second kinematic pass to fill the contact pair array
  contactData* cpu_pair_data = new contactData[contact_sum];
  contactData* gpu_pair_data;
  hipMalloc(&gpu_pair_data, contact_sum*sizeof(contactData));
  hipMemcpy(gpu_pair_data, cpu_pair_data, contact_sum*sizeof(contactData), hipMemcpyHostToDevice);

  kinematic2ndPass<<<1,k_n,0,kStream>>>(k_pos,k_n,gpu_offset_arr,k_num_arr,tolerance,k_radius,gpu_pair_data);

  hipMemcpy(cpu_pair_data, gpu_pair_data, contact_sum*sizeof(contactData), hipMemcpyDeviceToHost);

  //std::cout<<"contact pair num: "<<contact_sum<<std::endl;
  //std::cout<<"i: "<<cpu_pair_data[0].contact_pair.x<<", j:"<<cpu_pair_data[0].contact_pair.y<<std::endl;


  // share data through a common ThreadManager instance
  threadManager->contact_pair = cpu_pair_data;
  threadManager->contact_pair_n = contact_sum;

  threadManager->offset = offset_arr;
  threadManager->offset_n = k_n;

  hipDeviceSynchronize();
}

// *----------------------------------------
// Dynamic kernals
__global__ void dynamicTestKernel(){
    printf("dynamic run on GPU \n");
}

__global__ void dynamicPass(contactData* gpu_pair_data, int gpu_pair_n, vector3* gpu_pos, vector3* gpu_vel, vector3* gpu_acc, float radius){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= gpu_pair_n){return;}

  float dir_x = gpu_pos[gpu_pair_data[idx].contact_pair.x].x - gpu_pos[gpu_pair_data[idx].contact_pair.y].x;
  float dir_y = gpu_pos[gpu_pair_data[idx].contact_pair.x].y - gpu_pos[gpu_pair_data[idx].contact_pair.y].y;
  float dir_z = gpu_pos[gpu_pair_data[idx].contact_pair.x].z - gpu_pos[gpu_pair_data[idx].contact_pair.y].z;

  float dist2 = dir_x * dir_x + dir_y * dir_y + dir_z * dir_z;
  if(dist2 < radius * radius)
  {
    // TODO
  }


  __syncthreads();
}



// END of Dynamic kernels
// *----------------------------------------


void DynamicThread::dInitialize(float radius,vector3* pos, vector3* vel, vector3* acc, int n)
{
  d_n = n;
  d_radius = radius;
  hipSetDevice(this->device_id);
  hipMalloc(&d_pos, d_n*sizeof(vector3));
  hipMalloc(&d_vel, d_n*sizeof(vector3));
  hipMalloc(&d_acc, d_n*sizeof(vector3));

  hipMemcpy(d_pos, pos, d_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_vel, vel, d_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_acc, acc, d_n*sizeof(vector3), hipMemcpyHostToDevice);
}

void DynamicThread::doDynamicStep()
{
  hipSetDevice(this->device_id);
  //dynamicTestKernel<<<1, 1, 0, dStream>>>();

  // retrieve contact pair data from the ThreadManager
  contactData* cpu_pair = threadManager->contact_pair;
  int cpu_pair_n = threadManager->contact_pair_n;
  
  int* cpu_offset = threadManager->offset;
  int cpu_offset_n = threadManager->offset_n;

  // copy data to the Dynamic GPU
  contactData* gpu_pair_data;
  hipMalloc(&gpu_pair_data, cpu_pair_n*sizeof(contactData));
  hipMemcpy(gpu_pair_data, cpu_pair, cpu_pair_n*sizeof(contactData), hipMemcpyHostToDevice);

  // calculate number of threads needed and number of block needed
  int num_thread = 64;
  int num_block = cpu_pair_n / num_thread + 1;

  dynamicPass<<<num_block, num_thread, 0, dStream>>>(gpu_pair_data, cpu_pair_n, d_pos, d_vel, d_acc, d_radius);

  hipDeviceSynchronize();
}