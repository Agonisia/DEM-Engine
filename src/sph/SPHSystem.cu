// SPH-DualGPU
// SPH system base class implementation source code

#include <sph/SPHSystem.cuh>
#include <kernel/cudaKernels.cuh>


void SPHSystem::initialize(float radius,vector3 *pos, int n) {

  shared_data = new ExchangeData();

  m_pos = pos;
  m_n = n;

  vector3* vel = new vector3[n];
  vector3* acc = new vector3[n];

  GpuManager* gm = new GpuManager(2);


  // initialize two threads for Dual GPU computation
  kt = new KinematicTread(gm,shared_data);
  kt->kInitialize(radius, pos, vel, acc, n);
  dt = new DynamicThread(gm,shared_data);
  dt->dInitialize(radius, pos, vel, acc, n);
}

void SPHSystem::doStepDynamics(float time_step) {
  
  kt->doKinematicStep();
  dt->doDynamicStep();
}

void SPHSystem::printCSV(std::string filename) {
  // create file
  std::ofstream csvFile(filename);

  csvFile << "x_pos,y_pos,z_pos" << std::endl;

  // write particle data into csv file
  for (int i = 0; i < m_n; i++) {
    csvFile << m_pos[i].x << "," << m_pos[i].y << "," << m_pos[i].z
            << std::endl;
  }

  csvFile.close();
}


void KinematicTread::kInitialize(float radius,vector3* pos, vector3* vel, vector3* acc, int n)
{
  hipSetDevice(streamInfo.device);
  k_n = n;
  k_radius = radius;

  hipMalloc(&k_pos, k_n*sizeof(vector3));
  hipMalloc(&k_vel, k_n*sizeof(vector3));
  hipMalloc(&k_acc, k_n*sizeof(vector3));

  hipMemcpy(k_pos, pos, k_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(k_vel, vel, k_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(k_acc, acc, k_n*sizeof(vector3), hipMemcpyHostToDevice);
}

void KinematicTread::doKinematicStep()
{
  hipSetDevice(streamInfo.device);
  float tolerance = 0.05;
  //kinematicTestKernel<<<1, 1, 0, kStream>>>();

  // for each step, the kinematic thread needs to do two passes
  // first pass - look for 'number' of potential contacts
  // crate an array to store number of valid potential contacts
  int* cpu_num_arr = new int[k_n];
  for(int i = 0; i<k_n;i++)
  {
    cpu_num_arr[i] = -1;
  }
  int* k_num_arr;
  hipMalloc(&k_num_arr, k_n*sizeof(int));
  hipMemcpy(k_num_arr, cpu_num_arr, k_n*sizeof(int), hipMemcpyHostToDevice);

  // first kinematic pass to calculate offset array
  kinematic1stPass<<<1,k_n,0,streamInfo.stream>>>(k_pos, k_n, tolerance, k_radius, k_num_arr);

  hipDeviceSynchronize();

  hipMemcpy(cpu_num_arr, k_num_arr, k_n*sizeof(int), hipMemcpyDeviceToHost);

  // calculate the offset array
  int cur_idx = 0;
  int* offset_arr = new int[k_n];
  for(int i = 0; i<k_n;i++)
  {
    offset_arr[i] = cur_idx;
    cur_idx = cur_idx + cpu_num_arr[i];
  }

  int* gpu_offset_arr;
  hipMalloc(&gpu_offset_arr, k_n*sizeof(int));
  hipMemcpy(gpu_offset_arr, offset_arr, k_n*sizeof(int), hipMemcpyHostToDevice);

  // calculate total number of contact
  int contact_sum = 0;

  for(int i = 0; i < k_n; i++)
  {
    contact_sum = contact_sum + cpu_num_arr[i];
  }

  // second kinematic pass to fill the contact pair array
  contactData* cpu_pair_data = new contactData[contact_sum];
  contactData* gpu_pair_data;
  hipMalloc(&gpu_pair_data, contact_sum*sizeof(contactData));
  hipMemcpy(gpu_pair_data, cpu_pair_data, contact_sum*sizeof(contactData), hipMemcpyHostToDevice);

  kinematic2ndPass<<<1,k_n,0,streamInfo.stream>>>(k_pos,k_n,gpu_offset_arr,k_num_arr,tolerance,k_radius,gpu_pair_data);

  hipMemcpy(cpu_pair_data, gpu_pair_data, contact_sum*sizeof(contactData), hipMemcpyDeviceToHost);

  //std::cout<<"contact pair num: "<<contact_sum<<std::endl;
  //std::cout<<"i: "<<cpu_pair_data[0].contact_pair.x<<", j:"<<cpu_pair_data[0].contact_pair.y<<std::endl;


  // share data through a common ThreadManager instance
  k_shared_data->contact_pair = cpu_pair_data;
  k_shared_data->contact_pair_n = contact_sum;

  k_shared_data->offset = offset_arr;
  k_shared_data->offset_n = k_n;

  hipDeviceSynchronize();
}

void DynamicThread::dInitialize(float radius,vector3* pos, vector3* vel, vector3* acc, int n)
{
  hipSetDevice(streamInfo.device);
  d_n = n;
  d_radius = radius;

  hipMalloc(&d_pos, d_n*sizeof(vector3));
  hipMalloc(&d_vel, d_n*sizeof(vector3));
  hipMalloc(&d_acc, d_n*sizeof(vector3));

  hipMemcpy(d_pos, pos, d_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_vel, vel, d_n*sizeof(vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_acc, acc, d_n*sizeof(vector3), hipMemcpyHostToDevice);
}

void DynamicThread::doDynamicStep()
{
  hipSetDevice(streamInfo.device);
  //dynamicTestKernel<<<1, 1, 0, dStream>>>();

  // retrieve contact pair data from the ThreadManager
  contactData* cpu_pair = d_shared_data->contact_pair;
  int cpu_pair_n = d_shared_data->contact_pair_n;
  
  int* cpu_offset = d_shared_data->offset;
  int cpu_offset_n = d_shared_data->offset_n;

  // copy data to the Dynamic GPU
  contactData* gpu_pair_data;
  hipMalloc(&gpu_pair_data, cpu_pair_n*sizeof(contactData));
  hipMemcpy(gpu_pair_data, cpu_pair, cpu_pair_n*sizeof(contactData), hipMemcpyHostToDevice);

  // calculate number of threads needed and number of block needed
  int num_thread = 64;
  int num_block = cpu_pair_n / num_thread + 1;

  dynamicPass<<<num_block, num_thread, 0, streamInfo.stream>>>(gpu_pair_data, cpu_pair_n, d_pos, d_vel, d_acc, d_radius);

  hipDeviceSynchronize();
}